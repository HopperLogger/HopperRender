#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include "opticalFlowCalc.cuh"

// Kernel that blurs a frame
template <typename T>
__global__ void blurFrameKernel(const T* frameArray, T* blurredFrameArray, 
								const unsigned char kernelSize, const unsigned char chacheSize, const unsigned char boundsOffset, 
								const unsigned char avgEntriesPerThread, const unsigned short remainder, const char lumStart,
								const unsigned char lumEnd, const unsigned short lumPixelCount, const char chromStart, 
								const unsigned char chromEnd, const unsigned short chromPixelCount, const unsigned short dimY, const unsigned short dimX) {
	// Shared memory for the frame to prevent multiple global memory accesses
	extern __shared__ unsigned short sharedFrameArray[];

	// Current entry to be computed by the thread
	const unsigned short cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned short cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned char cz = blockIdx.z;

	// Check if the current thread is supposed to perform calculations
	if (cz == 1 && (cy >= dimY / 2 || cx >= dimX)) {
		return;
	}

	const unsigned short trX = blockIdx.x * blockDim.x;
	const unsigned short trY = blockIdx.y * blockDim.y;
	unsigned char offsetX;
	unsigned char offsetY;

    // Calculate the number of entries to fill for this thread
    const unsigned short threadIndex = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned char entriesToFill = avgEntriesPerThread + (threadIndex < remainder ? 1 : 0);

    // Calculate the starting index for this thread
    unsigned short startIndex = 0;
    for (unsigned short i = 0; i < threadIndex; ++i) {
        startIndex += avgEntriesPerThread + (i < remainder ? 1 : 0);
    }

    // Fill the shared memory for this thread
    for (unsigned short i = 0; i < entriesToFill; ++i) {
		offsetX = (startIndex + i) % chacheSize;
		offsetY = (startIndex + i) / chacheSize;
		if ((trY - boundsOffset + offsetY) < dimY && (trX - boundsOffset + offsetX) < dimX) {
			sharedFrameArray[startIndex + i] = frameArray[cz * dimY * dimX + (trY - boundsOffset + offsetY) * dimX + (trX - boundsOffset + offsetX)];
		} else {
			sharedFrameArray[startIndex + i] = 0;
		}
	}

    // Ensure all threads have finished loading before continuing
    __syncthreads();

	// Calculate the x and y boundaries of the kernel
	unsigned int blurredPixel = 0;

	// Collect the sum of the surrounding pixels
	// Y-Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		for (char y = lumStart; y < lumEnd; y++) {
			for (char x = lumStart; x < lumEnd; x++) {
				if ((cy + y) < dimY && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset + y) * chacheSize + threadIdx.x + boundsOffset + x];
				} else {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset) * chacheSize + threadIdx.x + boundsOffset];
				}
			}
		}
		blurredPixel /= lumPixelCount;
		blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
	// U/V-Channel
	} else if (cz == 1 && cy < dimY / 2 && cx < dimX) {
		for (char y = chromStart; y < chromEnd; y++) {
			for (char x = chromStart; x < chromEnd; x++) {
				if ((cy + y) < dimY / 2 && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset + y) * chacheSize + threadIdx.x + boundsOffset + x * 2];
				} else {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset) * chacheSize + threadIdx.x + boundsOffset];
				}
			}
		}
		blurredPixel /= chromPixelCount;
		blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
	}
}

// Kernel that sets the initial offset array
__global__ void setInitialOffset(int* offsetArray, const unsigned int numLayers, const unsigned int lowDimY, 
								 const unsigned int lowDimX, const unsigned int layerIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = blockIdx.z;

	if (cy < lowDimY && cx < lowDimX) {
		switch (cz) {
			// Set the X direction layer 1 to a -2 offset
			case 0:
				offsetArray[layerIdxOffset + cy * lowDimX + cx] = -2;
				return;
			// Set the X direction layer 2 to a -1 offset
			case 1:
				offsetArray[2 * layerIdxOffset + cy * lowDimX + cx] = -1;
				return;
			// Set the X direction layer 3 to a +1 offset
			case 2:
				offsetArray[3 * layerIdxOffset + cy * lowDimX + cx] = 1;
				return;
			// Set the X direction layer 4 to a +2 offset
			case 3:
				offsetArray[4 * layerIdxOffset + cy * lowDimX + cx] = 2;
				return;
		}
	}
}

// Helper kernel for the calcDeltaSums kernel
__device__ void warpReduce8x8(volatile unsigned int* partial_sums, int tIdx) {
	partial_sums[tIdx] += partial_sums[tIdx + 32];
	partial_sums[tIdx] += partial_sums[tIdx + 16];
	partial_sums[tIdx] += partial_sums[tIdx + 8];
	partial_sums[tIdx] += partial_sums[tIdx + 4];
	partial_sums[tIdx] += partial_sums[tIdx + 2];
	partial_sums[tIdx] += partial_sums[tIdx + 1];
}

// Helper kernel for the calcDeltaSums kernel
__device__ void warpReduce4x4(volatile unsigned int* partial_sums, int tIdx) {
	partial_sums[tIdx] += partial_sums[tIdx + 16];
	partial_sums[tIdx] += partial_sums[tIdx + 8];
	partial_sums[tIdx] += partial_sums[tIdx + 2];
	partial_sums[tIdx] += partial_sums[tIdx + 1];
}

// Helper kernel for the calcDeltaSums kernel
__device__ void warpReduce2x2(volatile unsigned int* partial_sums, int tIdx) {
	partial_sums[tIdx] += partial_sums[tIdx + 8];
	partial_sums[tIdx] += partial_sums[tIdx + 1];
}

// Kernel that sums up all the pixel deltas of each window
template <typename T>
__global__ void calcDeltaSums(unsigned int* summedUpDeltaArray, const T* frame1, const T* frame2,
							  const int* offsetArray, const unsigned int layerIdxOffset, const unsigned int directionIdxOffset,
						      const unsigned int dimY, const unsigned int dimX, const unsigned int lowDimY, const unsigned int lowDimX,
							  const unsigned int windowDim, const float resolutionScalar) {
	// Handle used to synchronize all threads
	auto g = cooperative_groups::this_thread_block();

	// Shared memory for the partial sums of the current block
	extern __shared__ unsigned int partial_sums[];

	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = blockIdx.z;
	const unsigned int tIdx = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned int layerOffset = blockIdx.z * lowDimY * lowDimX; // Offset to index the layer of the current thread
	const unsigned int scaledCx = static_cast<unsigned int>(static_cast<float>(cx) * resolutionScalar); // The X-Index of the current thread in the input frames
	const unsigned int scaledCy = static_cast<unsigned int>(static_cast<float>(cy) * resolutionScalar); // The Y-Index of the current thread in the input frames
	const unsigned int threadIndex2D = cy * lowDimX + cx; // Standard thread index without Z-Dim

	if (cy < lowDimY && cx < lowDimX) {
		// Calculate the image delta
		int offsetX = -offsetArray[layerOffset + threadIndex2D];
		int offsetY = -offsetArray[directionIdxOffset + layerOffset + threadIndex2D];
		int newCx = scaledCx + offsetX;
		int newCy = scaledCy + offsetY;

		// Window size of 1x1
		if (windowDim == 1) {
			summedUpDeltaArray[cz * layerIdxOffset + cy * lowDimX + cx] = (newCy < 0 || newCx < 0 || newCy >= dimY || newCx >= dimX) ? 0 : 
				abs(frame1[newCy * dimX + newCx] - frame2[scaledCy * dimX + scaledCx]);
		    return;
		// All other window sizes
		} else {
			partial_sums[tIdx] = (newCy < 0 || newCx < 0 || newCy >= dimY || newCx >= dimX) ? 0 : 
						abs(frame1[newCy * dimX + newCx] - frame2[scaledCy * dimX + scaledCx]);
		}
		
		__syncthreads();

		// Sum up the remaining pixels for the current window
		for (int s = (blockDim.y * blockDim.x) >> 1; s > 32; s >>= 1) {
			if (tIdx < s) {
				partial_sums[tIdx] += partial_sums[tIdx + s];
			}
			__syncthreads();
		}

		// Loop over the remaining values
		// Window size of 8x8 or larger
		if (windowDim >= 8) {
			if (tIdx < 32) {
				warpReduce8x8(partial_sums, tIdx);
			}
		// Window size of 4x4
		} else if (windowDim == 4) {
			// Top 4x4 Blocks
			if (threadIdx.y < 2) {
				warpReduce4x4(partial_sums, tIdx);
			// Bottom 4x4 Blocks
			} else if (threadIdx.y >= 4 && threadIdx.y < 6) {
				warpReduce4x4(partial_sums, tIdx);
			}
		// Window size of 2x2
		} else if (windowDim == 2) {
			if ((threadIdx.y & 1) == 0) {
				warpReduce2x2(partial_sums, tIdx);
			}
		}
		
		// Sync all threads
		g.sync();

		// Sum up the results of all blocks
		if ((windowDim >= 8 && tIdx == 0) || 
			(windowDim == 4 && (tIdx == 0 || tIdx == 4 || tIdx == 32 || tIdx == 36)) || 
			(windowDim == 2 && ((tIdx & 1) == 0 && (threadIdx.y & 1) == 0))) {
			const unsigned int windowIndexX = cx / windowDim;
			const unsigned int windowIndexY = cy / windowDim;
			atomicAdd(&summedUpDeltaArray[cz * layerIdxOffset + (windowIndexY * windowDim) * lowDimX + (windowIndexX * windowDim)], partial_sums[tIdx]);
		}
	}
}

// Kernel that normalizes all the pixel deltas of each window
__global__ void normalizeDeltaSums(const unsigned int* summedUpDeltaArray, unsigned char* globalLowestLayerArray,
                                   const int* offsetArray, const unsigned int windowDim, int numPixels,
								   const unsigned int directionIdxOffset, const unsigned int layerIdxOffset, 
								   const unsigned int numLayers, const unsigned int lowDimY, const unsigned int lowDimX) {
	// Allocate shared memory to share values across layers
	__shared__ float normalizedDeltaArray[5 * 8 * 8 * 4];
	
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const unsigned int threadIndex2D = cy * lowDimX + cx; // Standard thread index without Z-Dim
	bool isWindowRepresent = cy % windowDim == 0 && cx % windowDim == 0;

	// Check if the thread is a window represent
	if (isWindowRepresent) {
		// Get the current window information
		const int offsetX = offsetArray[cz * layerIdxOffset + threadIndex2D];
		const int offsetY = offsetArray[directionIdxOffset + cz * layerIdxOffset + threadIndex2D];

		// Calculate the not overlapping pixels
		int overlapX;
		int overlapY;

		// Calculate the number of not overlapping pixels
		if ((cx + windowDim + abs(offsetX) > lowDimX) || (cx - offsetX < 0)) {
			overlapX = abs(offsetX);
		} else {
			overlapX = 0;
		}

		if ((cy + windowDim + abs(offsetY) > lowDimY) || (cy - offsetY < 0)) {
			overlapY = abs(offsetY);
		} else {
			overlapY = 0;
		}

		const int numNotOverlappingPixels = overlapY * overlapX;
		numPixels -= numNotOverlappingPixels;
		numPixels = max(numPixels, 1);

		// Normalize the summed up delta
		normalizedDeltaArray[cz * 8 * 8 + threadIdx.y * 8 + threadIdx.x] = static_cast<float>(summedUpDeltaArray[cz * layerIdxOffset + threadIndex2D]) / static_cast<float>(numPixels);
	}

	// Wait for all threads to finish filling the values
	__syncthreads();

	// Find the layer with the lowest value
	if (cz == 0 && isWindowRepresent) {
		unsigned char lowestLayer = 0;

		for (unsigned char z = 1; z < numLayers; ++z) {
			if (normalizedDeltaArray[z * 8 * 8 + threadIdx.y * 8 + threadIdx.x] < 
				normalizedDeltaArray[lowestLayer * 8 * 8 + threadIdx.y * 8 + threadIdx.x]) {
				lowestLayer = z;
			}
		}

		globalLowestLayerArray[threadIndex2D] = lowestLayer;
	}
}

// Kernel that adjusts the offset array based on the comparison results
__global__ void adjustOffsetArray(int* offsetArray, const unsigned char* globalLowestLayerArray, unsigned char* statusArray,
								  const unsigned int windowDim, const unsigned int directionIdxOffset, const unsigned int layerIdxOffset, 
								  const unsigned int numLayers, const unsigned int lowDimY, const unsigned int lowDimX, const bool lastRun) {

	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int threadIndex2D = cy * lowDimX + cx; // Standard thread index without Z-Dim

	/*
	* Status Array Key:
	* 0: Find the initial x direction
	* 1: Find extended positive x direction
	* 2: Find extended negative x direction
	* 3: Find the initial y direction
	* 4: Find extended positive y direction
	* 5: Find extended negative y direction
	* 6: Search complete
	*/

	if (cy < lowDimY && cx < lowDimX) {
		const unsigned char currentStatus = statusArray[threadIndex2D];

		// We are done searching and we only need to do cleanup on the last run, so we exit here
		if (currentStatus == 6 && !lastRun) {
			return;
		}

		// We only need the lowestLayer if we are still searching
		unsigned char lowestLayer = 0;
		if (currentStatus != 6) {
			const unsigned int wx = (cx / windowDim) * windowDim;
			const unsigned int wy = (cy / windowDim) * windowDim;
			lowestLayer = globalLowestLayerArray[wy * lowDimX + wx];
		}

		int currX;
		int currY;

		// If this is the last run, we need to adjust the offset array for the next iteration
		if (lastRun) {
			switch (currentStatus) {
				// We are still trying to find the perfect x direction
				case 0:
				case 1:
				case 2:
					currX = offsetArray[lowestLayer * layerIdxOffset + threadIndex2D];

					// Shift the X direction layer 0 to the ideal X direction
					offsetArray[threadIndex2D] = currX;
					// Shift the X direction layer 1 by -2
					offsetArray[layerIdxOffset + threadIndex2D] = currX - 2;
					// Shift the X direction layer 2 by -1
					offsetArray[2 * layerIdxOffset + threadIndex2D] = currX - 1;
					// Shift the X direction layer 3 by +1
					offsetArray[3 * layerIdxOffset + threadIndex2D] = currX + 1;
					// Shift the X direction layer 4 by +2
					offsetArray[4 * layerIdxOffset + threadIndex2D] = currX + 2;
					return;

				// We are still trying to find the perfect y direction
				case 3:
				case 4:
				case 5:
					currX = offsetArray[threadIndex2D];
					currY = offsetArray[directionIdxOffset + lowestLayer * layerIdxOffset + threadIndex2D];

					// Set all Y direction layers to the ideal Y direction
					for (unsigned int z = 0; z < numLayers; z++) {
						offsetArray[directionIdxOffset + z * layerIdxOffset + threadIndex2D] = currY;
					}

					// Shift the X direction layer 1 by -2
					offsetArray[layerIdxOffset + threadIndex2D] = currX - 2;
					// Shift the X direction layer 2 by -1
					offsetArray[2 * layerIdxOffset + threadIndex2D] = currX - 1;
					// Shift the X direction layer 3 by +1
					offsetArray[3 * layerIdxOffset + threadIndex2D] = currX + 1;
					// Shift the X direction layer 4 by +2
					offsetArray[4 * layerIdxOffset + threadIndex2D] = currX + 2;
					return;

				// Search completed, so we adjust the offset array for the next run
				default:
					currX = offsetArray[threadIndex2D];
					currY = offsetArray[directionIdxOffset + threadIndex2D];

					// Set all Y direction layers to the ideal Y direction
					for (unsigned int z = 1; z < numLayers; z++) {
						offsetArray[directionIdxOffset + z * layerIdxOffset + threadIndex2D] = currY;
					}

					// Shift the X direction layer 1 by -2
					offsetArray[layerIdxOffset + threadIndex2D] = currX - 2;
					// Shift the X direction layer 2 by -1
					offsetArray[2 * layerIdxOffset + threadIndex2D] = currX - 1;
					// Shift the X direction layer 3 by +1
					offsetArray[3 * layerIdxOffset + threadIndex2D] = currX + 1;
					// Shift the X direction layer 4 by +2
					offsetArray[4 * layerIdxOffset + threadIndex2D] = currX + 2;
					return;
			}
			return;
		}

		// If we are still calculating, adjust the offset array based on the current status and lowest layer
		int idealX;
		int idealY;
		switch (currentStatus) {
			/*
			* X - DIRECTION
			*/
			// Find the initial x direction
			case 0:
				switch (lowestLayer) {
					// If the lowest layer is 0, no x direction is needed -> continue to y direction
					case 0:
						statusArray[threadIndex2D] = 3;
						currX = offsetArray[threadIndex2D];
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						for (int z = 0; z < numLayers; z++) {
							offsetArray[z * layerIdxOffset + threadIndex2D] = currX;
						}
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY + 2;
						return;
					
					// If the lowest layer is 1 -> continue moving in the negative x direction
					case 1:
						statusArray[threadIndex2D] = 2;
						currX = offsetArray[layerIdxOffset + threadIndex2D];
						offsetArray[threadIndex2D] = currX;
						offsetArray[layerIdxOffset + threadIndex2D] = currX - 1;
						offsetArray[2 * layerIdxOffset + threadIndex2D] = currX - 2;
						offsetArray[3 * layerIdxOffset + threadIndex2D] = currX - 3;
						offsetArray[4 * layerIdxOffset + threadIndex2D] = currX - 4;
						return;

					// If the lowest layer is 2, ideal x direction found -> continue to y direction
					case 2:
						statusArray[threadIndex2D] = 3;
						currX = offsetArray[2 * layerIdxOffset + threadIndex2D];
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						offsetArray[threadIndex2D] = currX;
						offsetArray[layerIdxOffset + threadIndex2D] = currX;
						offsetArray[3 * layerIdxOffset + threadIndex2D] = currX;
						offsetArray[4 * layerIdxOffset + threadIndex2D] = currX;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY + 2;
						return;

					// If the lowest layer is 3, ideal x direction found -> continue to y direction
					case 3:
						statusArray[threadIndex2D] = 3;
						currX = offsetArray[3 * layerIdxOffset + threadIndex2D];
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						offsetArray[threadIndex2D] = currX;
						offsetArray[layerIdxOffset + threadIndex2D] = currX;
						offsetArray[2 * layerIdxOffset + threadIndex2D] = currX;
						offsetArray[4 * layerIdxOffset + threadIndex2D] = currX;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY + 2;
						return;

					// If the lowest layer is 4 -> continue moving in the positive x direction
					case 4:
						statusArray[threadIndex2D] = 1;
						currX = offsetArray[4 * layerIdxOffset + threadIndex2D];
						offsetArray[threadIndex2D] = currX + 4;
						offsetArray[layerIdxOffset + threadIndex2D] = currX + 3;
						offsetArray[2 * layerIdxOffset + threadIndex2D] = currX + 2;
						offsetArray[3 * layerIdxOffset + threadIndex2D] = currX + 1;
						return;
				}
				return;

			// Find extended positive x direction
			case 1:
				switch (lowestLayer) {
					// If the lowest layer is 0 -> continue moving in x direction
					case 0:
						currX = offsetArray[threadIndex2D];
						offsetArray[threadIndex2D] = currX + 4;
						offsetArray[layerIdxOffset + threadIndex2D] = currX + 3;
						offsetArray[2 * layerIdxOffset + threadIndex2D] = currX + 2;
						offsetArray[3 * layerIdxOffset + threadIndex2D] = currX + 1;
						offsetArray[4 * layerIdxOffset + threadIndex2D] = currX;
						return;

					// If the lowest layer is not 0, no x further direction is needed -> continue to y direction
					default:
						statusArray[threadIndex2D] = 3;
						idealX = offsetArray[lowestLayer * layerIdxOffset + threadIndex2D];
						for (unsigned int z = 0; z < numLayers; z++) {
							offsetArray[z * layerIdxOffset + threadIndex2D] = idealX;
						}
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY + 2;
						return;
				}
				return;

			// Find extended negative x direction
			case 2:
				switch (lowestLayer) {
					// If the lowest layer is not 4, no x further direction is needed -> continue to y direction
					case 0:
					case 1:
					case 2:
					case 3:
						statusArray[threadIndex2D] = 3;
						idealX = offsetArray[lowestLayer * layerIdxOffset + threadIndex2D];
						for (unsigned int z = 0; z < numLayers; z++) {
							offsetArray[z * layerIdxOffset + threadIndex2D] = idealX;
						}
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY + 2;
						return;

					// If the lowest layer is 4 -> continue moving in x direction
					case 4:
						currX = offsetArray[4 * layerIdxOffset + threadIndex2D];
						offsetArray[threadIndex2D] = currX;
						offsetArray[layerIdxOffset + threadIndex2D] = currX - 1;
						offsetArray[2 * layerIdxOffset + threadIndex2D] = currX - 2;
						offsetArray[3 * layerIdxOffset + threadIndex2D] = currX - 3;
						offsetArray[4 * layerIdxOffset + threadIndex2D] = currX - 4;
						return;
				}
				return;

			/*
			* Y - DIRECTION
			*/
			// Find the initial y direction
			case 3:
				switch (lowestLayer) {
					// If the lowest layer is 0, 2, or 3, no y direction is needed -> we are done
					case 0:
					case 2:
					case 3:
						statusArray[threadIndex2D] = 6;
						if (lowestLayer != 0) {
							currY = offsetArray[directionIdxOffset + lowestLayer * layerIdxOffset + threadIndex2D];
							offsetArray[directionIdxOffset + threadIndex2D] = currY;
						}
						return;

					// If the lowest layer is 1 -> continue moving in the negative y direction
					case 1:
						statusArray[threadIndex2D] = 5;
						currY = offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = currY;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY - 3;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY - 4;
						return;

					// If the lowest layer is 4 -> continue moving in the positive y direction
					case 4:
						statusArray[threadIndex2D] = 4;
						currY = offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = currY + 4;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY + 3;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY + 2;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						return;
				}
				return;

			// Find extended positive y direction
			case 4:
				switch (lowestLayer) {
					// If the lowest layer is 0 -> continue moving in y direction
					case 0:
						currY = offsetArray[directionIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = currY + 4;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY + 3;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY + 2;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY + 1;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY;
						return;

					// If the lowest layer is not 0, no y further direction is needed -> we are done
					default:
						statusArray[threadIndex2D] = 6;
						idealY = offsetArray[directionIdxOffset + lowestLayer * layerIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = idealY;
						return;
				}
				return;

			// Find extended negative y direction
			case 5:
				switch (lowestLayer) {
					// If the lowest layer is not 4, no y further direction is needed -> we are done
					case 0:
					case 1:
					case 2:
					case 3:
						statusArray[threadIndex2D] = 6;
						idealY = offsetArray[directionIdxOffset + lowestLayer * layerIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = idealY;
						return;

					// If the lowest layer is 4 -> continue moving in y direction
					case 4:
						currY = offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D];
						offsetArray[directionIdxOffset + threadIndex2D] = currY;
						offsetArray[directionIdxOffset + layerIdxOffset + threadIndex2D] = currY - 1;
						offsetArray[directionIdxOffset + 2 * layerIdxOffset + threadIndex2D] = currY - 2;
						offsetArray[directionIdxOffset + 3 * layerIdxOffset + threadIndex2D] = currY - 3;
						offsetArray[directionIdxOffset + 4 * layerIdxOffset + threadIndex2D] = currY - 4;
						return;
				}
				return;

			// Search is complete
			default:
				return;
		}
	}
}

// Kernel that translates a flow array from frame 1 to frame 2 into a flow array from frame 2 to frame 1
__global__ void flipFlowKernel(const int* flowArray12, int* flowArray21, const int lowDimY, const int lowDimX, 
							   const float resolutionDivider, const unsigned int directionIdxOffset,
							   const unsigned int layerIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	// Check if we are inside the flow array
	if (cy < lowDimY && cx < lowDimX) {
		// Get the current flow values
		const int x = flowArray12[cy * lowDimX + cx];
		const int y = flowArray12[directionIdxOffset + cy * lowDimX + cx];
		const int scaledX = static_cast<int>(static_cast<float>(x) * resolutionDivider);
		const int scaledY = static_cast<int>(static_cast<float>(y) * resolutionDivider);

		// Project the flow values onto the flow array from frame 2 to frame 1
		// X-Layer
		if (cz == 0 && (cy + scaledY) < lowDimY && (cy + scaledY) >= 0 && (cx + scaledX) < lowDimX && (cx + scaledX) >= 0) {
			flowArray21[(cy + scaledY) * lowDimX + cx + scaledX] = -x;
		// Y-Layer
		} else if (cz == 1 && (cy + scaledY) < lowDimY && (cy + scaledY) >= 0 && (cx + scaledX) < lowDimX && (cx + scaledX) >= 0) {
			flowArray21[layerIdxOffset + (cy + scaledY) * lowDimX + cx + scaledX] = -y;
		}
	}
}

// Kernel that blurs a flow array
__global__ void blurFlowKernel(const int* flowArray, int* blurredFlowArray, 
								const unsigned char kernelSize, const unsigned char chacheSize, const unsigned char boundsOffset, 
								const unsigned char avgEntriesPerThread, const unsigned short remainder, const char start,
								const unsigned char end, const unsigned short pixelCount, const unsigned short numLayers,
								const unsigned short lowDimY, const unsigned short lowDimX) {
	// Shared memory for the flow to prevent multiple global memory accesses
	extern __shared__ int sharedFlowArray[];

	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = blockIdx.z;

	// Current threadblock index
	const int trX = blockIdx.x * blockDim.x;
	const int trY = blockIdx.y * blockDim.y;
	unsigned char offsetX;
	unsigned char offsetY;

    // Calculate the number of entries to fill for this thread
    const unsigned short threadIndex = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned char entriesToFill = avgEntriesPerThread + (threadIndex < remainder ? 1 : 0);

    // Calculate the starting index for this thread
    unsigned short startIndex = 0;
    for (unsigned short i = 0; i < threadIndex; ++i) {
        startIndex += avgEntriesPerThread + (i < remainder ? 1 : 0);
    }

    // Fill the shared memory for this thread
    for (unsigned short i = 0; i < entriesToFill; ++i) {
		offsetX = (startIndex + i) % chacheSize;
		offsetY = (startIndex + i) / chacheSize;
		if ((trY - boundsOffset + offsetY) < lowDimY && (trX - boundsOffset + offsetX) < lowDimX) {
			sharedFlowArray[startIndex + i] = flowArray[cz * numLayers * lowDimY * lowDimX + (trY - boundsOffset + offsetY) * lowDimX + (trX - boundsOffset + offsetX)];
		} else {
			sharedFlowArray[startIndex + i] = 0;
		}
	}

    // Ensure all threads have finished loading before continuing
    __syncthreads();

	// Check if we are inside the flow array
	if (cy < lowDimY && cy >= 0 && cx < lowDimX && cx >= 0) {
		// Calculate the x and y boundaries of the kernel
		int blurredOffset = 0;

		// Collect the sum of the surrounding values
		for (char y = start; y < end; y++) {
			for (char x = start; x < end; x++) {
				if ((cy + y) < lowDimY && (cy + y) >= 0 && (cx + x) < lowDimX && (cx + x) >= 0) {
					blurredOffset += sharedFlowArray[(threadIdx.y + boundsOffset + y) * chacheSize + threadIdx.x + boundsOffset + x];
				} else {
					blurredOffset += sharedFlowArray[(threadIdx.y + boundsOffset) * chacheSize + threadIdx.x + boundsOffset];
				}
			}
		}
		blurredOffset /= pixelCount;
		blurredFlowArray[cz * lowDimY * lowDimX + cy * lowDimX + cx] = blurredOffset;
	}
	
}

// Kernel that cleans a flow array
__global__ void cleanFlowKernel(const int* flowArray, int* blurredFlowArray, 
								const unsigned short lowDimY, const unsigned short lowDimX) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	int offsetX = flowArray[cy * lowDimX + cx];
	int offsetY = flowArray[lowDimY * lowDimX + cy * lowDimX + cx];

    if (abs(offsetY) <= 2 && abs(offsetX <= 2)) {
		blurredFlowArray[cz * lowDimY * lowDimX + cy * lowDimX + cx] = flowArray[cz * lowDimY * lowDimX + cy * lowDimX + cx];
	}
}

// Kernel that warps a frame according to the offset array
template <typename T, typename S>
__global__ void warpFrameKernel(const T* frame1, const int* offsetArray, int* hitCount,
								S* warpedFrame, const float frameScalar, const unsigned int lowDimY, const unsigned int lowDimX,
								const unsigned int dimY, const int dimX, const float resolutionDivider,
								const unsigned int directionIdxOffset, const unsigned int scaledDimX, const unsigned int channelIdxOffset, 
								const unsigned int scaledChannelIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = blockIdx.z * blockDim.z + threadIdx.z;
	const bool shift = sizeof(T) == 1 && sizeof(S) == 2; // SDR && DirectOutput

	const int scaledCx = static_cast<int>(static_cast<float>(cx) * resolutionDivider); // The X-Index of the current thread in the offset array
	const int scaledCy = static_cast<int>(static_cast<float>(cy) * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		// Get the current offsets to use
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY && newCx >= 0 && newCx < dimX) {
			warpedFrame[newCy * scaledDimX + newCx] = static_cast<S>(frame1[cy * dimX + cx]) << (shift ? 8 : 0);
			atomicAdd(&hitCount[newCy * dimX + newCx], 1);
		}

	// U/V-Channel
	} else if (cz == 1 && cy < (dimY >> 1) && cx < dimX) {
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[(scaledCy << 1) * lowDimX + (scaledCx & ~1)]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + (scaledCy << 1) * lowDimX + (scaledCx & ~1)]) * frameScalar) >> 1;
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < (dimY >> 1) && newCx >= 0 && newCx < dimX) {
			// U-Channel
			if ((cx & 1) == 0) {
				warpedFrame[scaledChannelIdxOffset + newCy * scaledDimX + (newCx & ~1)] = static_cast<S>(frame1[channelIdxOffset + cy * dimX + cx]) << (shift ? 8 : 0);

			// V-Channel
			} else {
				warpedFrame[scaledChannelIdxOffset + newCy * scaledDimX + (newCx & ~1) + 1] = static_cast<S>(frame1[channelIdxOffset + cy * dimX + cx]) << (shift ? 8 : 0);
			}
		}
	}
}

// Kernel that removes artifacts from the warped frame
template <typename T, typename S>
__global__ void artifactRemovalKernel(const T* frame1, const int* hitCount, S* warpedFrame,
												 const unsigned int dimY, const unsigned int dimX, const int scaledDimX,
												  const unsigned int channelIdxOffset, const unsigned int scaledChannelIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const unsigned int threadIndex2D = cy * dimX + cx; // Standard thread index without Z-Dim
	const bool shift = sizeof(T) == 1 && sizeof(S) == 2; // SDR && DirectOutput

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		if (hitCount[threadIndex2D] != 1) {
			warpedFrame[cy * scaledDimX + cx] = static_cast<S>(frame1[threadIndex2D]) << (shift ? 8 : 0);
		}

	// U/V Channels
	} else if (cz == 1 && cy < (dimY >> 1) && cx < dimX) {
		if (hitCount[threadIndex2D] != 1) {
			warpedFrame[scaledChannelIdxOffset + cy * scaledDimX + cx] = static_cast<S>(frame1[channelIdxOffset + threadIndex2D]) << (shift ? 8 : 0);
		}
	}
}

// Kernel that blends warpedFrame1 to warpedFrame2
template <typename T>
__global__ void blendFrameKernel(const T* warpedFrame1, const T* warpedFrame2, unsigned short* outputFrame,
                                 const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                 const unsigned int dimX, const int scaledDimX, const unsigned int channelIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const bool isHDR = sizeof(T) == 2;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		outputFrame[cy * scaledDimX + cx] = 
			static_cast<unsigned short>(
				static_cast<float>(warpedFrame1[cy * dimX + cx]) * frame1Scalar + 
				static_cast<float>(warpedFrame2[cy * dimX + cx]) * frame2Scalar
			) << (isHDR ? 0 : 8);
	// U/V Channels
	} else if (cz == 1 && cy < (dimY >> 1) && cx < dimX) {
		outputFrame[dimY * scaledDimX + cy * scaledDimX + cx] = 
			static_cast<unsigned short>(
				static_cast<float>(warpedFrame1[channelIdxOffset + cy * dimX + cx]) * frame1Scalar + 
				static_cast<float>(warpedFrame2[channelIdxOffset + cy * dimX + cx]) * frame2Scalar
			) << (isHDR ? 0 : 8);
	}
}

// Kernel that places half of frame 1 over the outputFrame
template <typename T>
__global__ void insertFrameKernel(const T* frame1, unsigned short* outputFrame, const unsigned int dimY,
                                  const unsigned int dimX, const int scaledDimX, const unsigned int channelIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const bool isHDR = sizeof(T) == 2;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < (dimX >> 1)) {
		outputFrame[cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[cy * dimX + cx]) << (isHDR ? 0 : 8);
	// U/V Channels
	} else if (cz == 1 && cy < (dimY >> 1) && cx < (dimX >> 1)) {
		outputFrame[dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[channelIdxOffset + cy * dimX + cx]) << (isHDR ? 0 : 8);
	}
}

// Kernel that places frame 1 scaled down on the left side and the blendedFrame on the right side of the outputFrame
template <typename T>
__global__ void sideBySideFrameKernel(const T* frame1, const T* warpedFrame1, const T* warpedFrame2, unsigned short* outputFrame, 
									  const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                      const unsigned int dimX, const int scaledDimX, const unsigned int halfDimY, 
									  const unsigned int halfDimX, const unsigned int channelIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const unsigned int verticalOffset = dimY >> 2;
	const bool isYChannel = cz == 0 && cy < dimY && cx < dimX;
	const bool isUVChannel = cz == 1 && cy < halfDimY && cx < dimX;
	const bool isInLeftSideY = cy >= verticalOffset && cy < (verticalOffset + halfDimY) && cx < halfDimX;
	const bool isInRightSideY = cy >= verticalOffset && cy < (verticalOffset + halfDimY) && cx >= halfDimX && cx < dimX;
	const bool isInLeftSideUV = cy >= (verticalOffset >> 1) && cy < ((verticalOffset >> 1) + (dimY >> 2)) && cx < halfDimX;
	const bool isInRightSideUV = cy >= (verticalOffset >> 1) && cy < ((verticalOffset >> 1) + (dimY >> 2)) && cx >= halfDimX && cx < dimX;
	const bool isVChannel = (cx & 1) == 1;
	const bool isHDR = sizeof(T) == 2;
	unsigned short blendedFrameValue;

	// --- Blending ---
	// Y Channel
	if (isYChannel && isInRightSideY) {
		blendedFrameValue = 
			static_cast<unsigned short>(
				static_cast<float>(warpedFrame1[((cy - verticalOffset) << 1) * dimX + ((cx - halfDimX) << 1)]) * frame1Scalar + 
				static_cast<float>(warpedFrame2[((cy - verticalOffset) << 1) * dimX + ((cx - halfDimX) << 1)]) * frame2Scalar
			) << (isHDR ? 0 : 8);
	// U/V Channels
	} else if (isUVChannel && isInRightSideUV) {
		blendedFrameValue = 
			static_cast<unsigned short>(
				static_cast<float>(warpedFrame1[channelIdxOffset + 2 * (cy - (verticalOffset >> 1)) * dimX + ((cx - halfDimX) << 1) + isVChannel]) * frame1Scalar + 
				static_cast<float>(warpedFrame2[channelIdxOffset + 2 * (cy - (verticalOffset >> 1)) * dimX + ((cx - halfDimX) << 1) + isVChannel]) * frame2Scalar
			) << (isHDR ? 0 : 8);
	}

	// Y Channel
	if (isYChannel) {
		if (isInLeftSideY) {
			outputFrame[cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[((cy - verticalOffset) << 1) * dimX + (cx << 1)]) << (isHDR ? 0 : 8);
		} else if (isInRightSideY) {
			outputFrame[cy * scaledDimX + cx] = blendedFrameValue;
		} else {
			outputFrame[cy * scaledDimX + cx] = 0;
		}
	} else if (isUVChannel) {
		if (isInLeftSideUV) {
			outputFrame[dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[channelIdxOffset + ((cy - (verticalOffset >> 1)) << 1) * dimX + (cx << 1) + isVChannel]) << (isHDR ? 0 : 8);
		} else if (isInRightSideUV) {
			outputFrame[dimY * scaledDimX + cy * scaledDimX + cx] = blendedFrameValue;
		} else {
			outputFrame[dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(128) << 8;
		}
	}
}

// Kernel that creates an HSV flow image from the offset array
template <typename T>
__global__ void convertFlowToHSVKernel(const int* flowArray, unsigned short* outputFrame, const T* frame1,
                                       const float blendScalar, const unsigned int lowDimX, const unsigned int dimY, const unsigned int dimX, 
									   const float resolutionDivider, const unsigned int directionIdxOffset, const unsigned int scaledDimX,
									   const unsigned int scaledChannelIdxOffset) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const bool isHDR = sizeof(T) == 2;

	const unsigned int scaledCx = static_cast<unsigned int>(static_cast<float>(cx) * resolutionDivider); // The X-Index of the current thread in the offset array
	const unsigned int scaledCy = static_cast<unsigned int>(static_cast<float>(cy) * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Get the current flow values
	float x;
	float y;
	if (cz == 0) {
		x = flowArray[scaledCy * lowDimX + scaledCx];
		y = flowArray[directionIdxOffset + scaledCy * lowDimX + scaledCx];
	} else {
		x = flowArray[(scaledCy << 1) * lowDimX + scaledCx];
		y = flowArray[directionIdxOffset + (scaledCy << 1) * lowDimX + scaledCx];
	}

	// RGB struct
	struct RGB {
		int r, g, b;
	};

	// Calculate the angle in radians
	const float angle_rad = std::atan2(y, x);

	// Convert radians to degrees
	float angle_deg = angle_rad * (180.0f / 3.14159265359f);

	// Ensure the angle is positive
	if (angle_deg < 0) {
		angle_deg += 360.0f;
	}

	// Normalize the angle to the range [0, 360]
	angle_deg = fmodf(angle_deg, 360.0f);
	if (angle_deg < 0) {
		angle_deg += 360.0f;
	}

	// Map the angle to the hue value in the HSV model
	const float hue = angle_deg / 360.0f;

	// Convert HSV to RGB
	const int h_i = static_cast<int>(hue * 6.0f);
	const float f = hue * 6.0f - h_i;
	const float q = 1.0f - f;

	RGB rgb;
	switch (h_i % 6) {
		case 0: rgb = { static_cast<int>(255), static_cast<int>(f * 255), 0 }; break;
		case 1: rgb = { static_cast<int>(q * 255), static_cast<int>(255), 0 }; break;
		case 2: rgb = { 0, static_cast<int>(255), static_cast<int>(f * 255) }; break;
		case 3: rgb = { 0, static_cast<int>(q * 255), static_cast<int>(255) }; break;
		case 4: rgb = { static_cast<int>(f * 255), 0, static_cast<int>(255) }; break;
		case 5: rgb = { static_cast<int>(255), 0, static_cast<int>(q * 255) }; break;
		default: rgb = { 0, 0, 0 }; break;
	}

	// Prevent random colors when there is no flow
	if (fabsf(x) < 1.0f && fabsf(y) < 1.0f) {
		rgb = { 0, 0, 0 };
	}

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		outputFrame[cy * scaledDimX + cx] = isHDR ?
			(static_cast<unsigned short>(
				(fmaxf(fminf(0.299f * rgb.r + 0.587f * rgb.g + 0.114f * rgb.b, 255.0f), 0.0f)) * blendScalar) << 8) + 
				frame1[cy * dimX + cx] * (1.0f - blendScalar)
			:
			static_cast<unsigned short>(
				(fmaxf(fminf(0.299f * rgb.r + 0.587f * rgb.g + 0.114f * rgb.b, 255.0f), 0.0f)) * blendScalar + 
				frame1[cy * dimX + cx] * (1.0f - blendScalar)
			) << 8;
	// U/V Channels
	} else if (cz == 1 && cy < (dimY >> 1) && cx < dimX) {
		// U Channel
		if ((cx & 1) == 0) {
			outputFrame[scaledChannelIdxOffset + cy * scaledDimX + (cx & ~1)] = 
				static_cast<unsigned short>(
					fmaxf(fminf(0.492f * (rgb.b - (0.299f * rgb.r + 0.587f * rgb.g + 0.114 * rgb.b)) + 128.0f, 255.0f), 0.0f)
				) << 8;
		// V Channel
		} else {
			outputFrame[scaledChannelIdxOffset + cy * scaledDimX + (cx & ~1) + 1] = 
				static_cast<unsigned short>(
					fmaxf(fminf(0.877f * (rgb.r - (0.299f * rgb.r + 0.587f * rgb.g + 0.114f * rgb.b)) + 128.0f, 255.0f), 0.0f)
				) << 8;
		}
	}
}

/*
* Translates a flow array from frame 1 to frame 2 into a flow array from frame 2 to frame 1
*/
void OpticalFlowCalc::flipFlow() const {
	// Reset the offset array
	m_offsetArray21.zero();

	// Launch kernel
	flipFlowKernel << <m_lowGrid16x16x1, m_threads16x16x2 >> > (m_offsetArray12.arrayPtrGPU, m_offsetArray21.arrayPtrGPU,
												            m_iLowDimY, m_iLowDimX, m_fResolutionDivider, m_iDirectionIdxOffset, m_iLayerIdxOffset);
}

/*
* Blurs the offset arrays
*
* @param kernelSize: Size of the kernel to use for the blur
*/
void OpticalFlowCalc::blurFlowArrays(const int kernelSize) const {
	const unsigned char boundsOffset = kernelSize >> 1;
	const unsigned char chacheSize = kernelSize + (boundsOffset << 1);
	const size_t sharedMemSize = chacheSize * chacheSize * sizeof(int);
	const unsigned short totalThreads = max(kernelSize * kernelSize, 1);
    const unsigned short totalEntries = chacheSize * chacheSize;
    const unsigned char avgEntriesPerThread = totalEntries / totalThreads;
	const unsigned short remainder = totalEntries % totalThreads;
	const char start = -(kernelSize >> 1);
	const unsigned char end = (kernelSize >> 1);
	const unsigned short pixelCount = (end - start) * (end - start);

	// Calculate the number of blocks needed
	const unsigned int NUM_BLOCKS_X = max(static_cast<int>(ceil(static_cast<double>(m_iLowDimX) / kernelSize)), 1);
	const unsigned int NUM_BLOCKS_Y = max(static_cast<int>(ceil(static_cast<double>(m_iLowDimY) / kernelSize)), 1);

	// Use dim3 structs for block and grid size
	dim3 gridBF(NUM_BLOCKS_X, NUM_BLOCKS_Y, 2);
	dim3 threadsBF(kernelSize, kernelSize, 1);

	// No need to blur the flow if the kernel size is less than 4
	if (kernelSize < 4) {
		// Offset12 X-Dir
		hipMemcpy(m_blurredOffsetArray12.arrayPtrGPU, m_offsetArray12.arrayPtrGPU, m_iLayerIdxOffset * sizeof(int), hipMemcpyDeviceToDevice);
		// Offset12 Y-Dir
		hipMemcpy(m_blurredOffsetArray12.arrayPtrGPU + m_iLayerIdxOffset, m_offsetArray12.arrayPtrGPU + m_iDirectionIdxOffset, m_iLayerIdxOffset * sizeof(int), hipMemcpyDeviceToDevice);
		// Offset21 X&Y-Dir
		hipMemcpy(m_blurredOffsetArray21.arrayPtrGPU, m_offsetArray21.arrayPtrGPU, m_offsetArray21.bytes, hipMemcpyDeviceToDevice);
	} else {
		// Create CUDA streams
		hipStream_t blurStream1, blurStream2;
		hipStreamCreate(&blurStream1);
		hipStreamCreate(&blurStream2);

		// Launch kernels
		blurFlowKernel << <gridBF, threadsBF, sharedMemSize, blurStream1 >> > (m_offsetArray12.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU, kernelSize, chacheSize, boundsOffset, avgEntriesPerThread, remainder, start, end, pixelCount, m_iNumLayers, m_iLowDimY, m_iLowDimX);
		blurFlowKernel << <gridBF, threadsBF, sharedMemSize, blurStream2 >> > (m_offsetArray21.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU, kernelSize, chacheSize, boundsOffset, avgEntriesPerThread, remainder, start, end, pixelCount, 1, m_iLowDimY, m_iLowDimX);
		//cleanFlowKernel << <m_lowGrid16x16x1, m_threads16x16x2, 0, blurStream1 >> > (m_offsetArray12.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU, m_iLowDimY, m_iLowDimX);
		//cleanFlowKernel << <m_lowGrid16x16x1, m_threads16x16x2, 0, blurStream2 >> > (m_offsetArray21.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU, m_iLowDimY, m_iLowDimX);


		// Synchronize streams to ensure completion
		hipStreamSynchronize(blurStream1);
		hipStreamSynchronize(blurStream2);

		// Clean up streams
		hipStreamDestroy(blurStream1);
		hipStreamDestroy(blurStream2);
	}
}

/*
* Template instantiation
*/
template __global__ void blurFrameKernel(const unsigned char* frameArray, unsigned char* blurredFrameArray,
		const unsigned char kernelSize, const unsigned char chacheSize,
		const unsigned char boundsOffset,
		const unsigned char avgEntriesPerThread,
		const unsigned short remainder, const char lumStart,
		const unsigned char lumEnd, const unsigned short lumPixelCount,
		const char chromStart, const unsigned char chromEnd,
		const unsigned short chromPixelCount, const unsigned short dimY,
		const unsigned short dimX);
template __global__ void blurFrameKernel(const unsigned short* frameArray, unsigned short* blurredFrameArray,
		const unsigned char kernelSize, const unsigned char chacheSize,
		const unsigned char boundsOffset,
		const unsigned char avgEntriesPerThread,
		const unsigned short remainder, const char lumStart,
		const unsigned char lumEnd, const unsigned short lumPixelCount,
		const char chromStart, const unsigned char chromEnd,
		const unsigned short chromPixelCount, const unsigned short dimY,
		const unsigned short dimX);
template __global__ void calcDeltaSums(unsigned int* summedUpDeltaArray, const unsigned char* frame1, const unsigned char* frame2,
							   const int* offsetArray, const unsigned int layerIdxOffset, const unsigned int directionIdxOffset,
						const unsigned int dimY, const unsigned int dimX, const unsigned int lowDimY, const unsigned int lowDimX, const unsigned int windowDim, const float resolutionScalar);
template __global__ void calcDeltaSums(unsigned int* summedUpDeltaArray, const unsigned short* frame1, const unsigned short* frame2,
							   const int* offsetArray, const unsigned int layerIdxOffset, const unsigned int directionIdxOffset,
						const unsigned int dimY, const unsigned int dimX, const unsigned int lowDimY, const unsigned int lowDimX, const unsigned int windowDim, const float resolutionScalar);

template __global__ void warpFrameKernel(
    const unsigned char* frame1, const int* offsetArray, int* hitCount,
    unsigned char* warpedFrame, const float frameScalar, const unsigned int lowDimY,
    const unsigned int lowDimX, const unsigned int dimY, const int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int channelIdxOffset,
    const unsigned int scaledChannelIdxOffset);
template __global__ void warpFrameKernel(
    const unsigned short* frame1, const int* offsetArray, int* hitCount,
    unsigned char* warpedFrame, const float frameScalar, const unsigned int lowDimY,
    const unsigned int lowDimX, const unsigned int dimY, const int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int channelIdxOffset,
    const unsigned int scaledChannelIdxOffset);
template __global__ void warpFrameKernel(
    const unsigned char* frame1, const int* offsetArray, int* hitCount,
    unsigned short* warpedFrame, const float frameScalar, const unsigned int lowDimY,
    const unsigned int lowDimX, const unsigned int dimY, const int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int channelIdxOffset,
    const unsigned int scaledChannelIdxOffset);
template __global__ void warpFrameKernel(
    const unsigned short* frame1, const int* offsetArray, int* hitCount,
    unsigned short* warpedFrame, const float frameScalar, const unsigned int lowDimY,
    const unsigned int lowDimX, const unsigned int dimY, const int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int channelIdxOffset,
    const unsigned int scaledChannelIdxOffset);

template __global__ void artifactRemovalKernel(const unsigned char* frame1, const int* hitCount, unsigned char* warpedFrame,
		      const unsigned int dimY, const unsigned int dimX,
		      const int scaledDimX, const unsigned int channelIdxOffset,
		      const unsigned int scaledChannelIdxOffset);
template __global__ void artifactRemovalKernel(const unsigned short* frame1, const int* hitCount, unsigned char* warpedFrame,
		      const unsigned int dimY, const unsigned int dimX,
		      const int scaledDimX, const unsigned int channelIdxOffset,
		      const unsigned int scaledChannelIdxOffset);
template __global__ void artifactRemovalKernel(const unsigned char* frame1, const int* hitCount, unsigned short* warpedFrame,
		      const unsigned int dimY, const unsigned int dimX,
		      const int scaledDimX, const unsigned int channelIdxOffset,
		      const unsigned int scaledChannelIdxOffset);
template __global__ void artifactRemovalKernel(const unsigned short* frame1, const int* hitCount, unsigned short* warpedFrame,
		      const unsigned int dimY, const unsigned int dimX,
		      const int scaledDimX, const unsigned int channelIdxOffset,
		      const unsigned int scaledChannelIdxOffset);

template __global__ void blendFrameKernel(const unsigned char* warpedFrame1, const unsigned char* warpedFrame2,
		 unsigned short* outputFrame, const float frame1Scalar,
		 const float frame2Scalar, const unsigned int dimY,
		 const unsigned int dimX, const int scaledDimX,
		 const unsigned int channelIdxOffset);
template __global__ void blendFrameKernel(const unsigned short* warpedFrame1, const unsigned short* warpedFrame2,
		 unsigned short* outputFrame, const float frame1Scalar,
		 const float frame2Scalar, const unsigned int dimY,
		 const unsigned int dimX, const int scaledDimX,
		 const unsigned int channelIdxOffset);

template __global__ void insertFrameKernel(const unsigned char* frame1, unsigned short* outputFrame,
		  const unsigned int dimY, const unsigned int dimX,
		  const int scaledDimX, const unsigned int channelIdxOffset);
template __global__ void insertFrameKernel(const unsigned short* frame1, unsigned short* outputFrame,
		  const unsigned int dimY, const unsigned int dimX,
		  const int scaledDimX, const unsigned int channelIdxOffset);

template __global__ void sideBySideFrameKernel(const unsigned char* frame1, const unsigned char* warpedFrame1, const unsigned char* warpedFrame2, unsigned short* outputFrame, 
									  const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                      const unsigned int dimX, const int scaledDimX, const unsigned int halfDimY, 
									  const unsigned int halfDimX, const unsigned int channelIdxOffset);
template __global__ void sideBySideFrameKernel(const unsigned short* frame1, const unsigned short* warpedFrame1, const unsigned short* warpedFrame2, unsigned short* outputFrame, 
									  const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                      const unsigned int dimX, const int scaledDimX, const unsigned int halfDimY, 
									  const unsigned int halfDimX, const unsigned int channelIdxOffset);

template __global__ void convertFlowToHSVKernel(
    const int* flowArray, unsigned short* outputFrame, const unsigned char* frame1,
    const float blendScalar, const unsigned int lowDimX,
    const unsigned int dimY, const unsigned int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int scaledChannelIdxOffset);
template __global__ void convertFlowToHSVKernel(
    const int* flowArray, unsigned short* outputFrame, const unsigned short* frame1,
    const float blendScalar, const unsigned int lowDimX,
    const unsigned int dimY, const unsigned int dimX,
    const float resolutionDivider, const unsigned int directionIdxOffset,
    const unsigned int scaledDimX, const unsigned int scaledChannelIdxOffset);