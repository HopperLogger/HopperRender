#include "hip/hip_runtime.h"
// opticalFlowCalc.cu

// CUDA libaries
#include "hip/hip_runtime.h"
#include ""

// Project Includes
#include "opticalFlowCalc.cuh"

// C++ libaries
#include <iostream>
#include <iomanip>
#include <chrono>
#include <vector>
#include <math.h>

// Kernal that calculates the absolute difference between two frames using the offset array
__global__ void calcImageDelta(unsigned char* frame1, unsigned char* frame2, unsigned char* imageDeltaArray, int* offsetArray, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;
	int cz = threadIdx.z;

	// Get the current offsets to use
	int offsetX = -offsetArray[cy * dimX + cx];
	int offsetY = -offsetArray[dimY * dimX + cy * dimX + cx];

	// Check if the thread is inside the frame (without offsets)
	if (cz < 3 && cy < dimY && cx < dimX) {
		// Current pixel is outside of frame
		if ((cy + offsetY < 0) || (cx + offsetX < 0) || (cy + offsetY > dimY) || (cx + offsetX > dimX)) {
			imageDeltaArray[cz * dimY * dimX + cy * dimX + cx] = 0;
		}
		// Current pixel is inside of frame
		else {
			imageDeltaArray[cz * dimY * dimX + cy * dimX + cx] = fabsf(frame1[cz * dimY * dimX + cy * dimX + cx + (offsetY * dimX + offsetX)] - frame2[cz * dimY * dimX + cy * dimX + cx]);
		}
	}
}

// Kernal that sums up all the pixel deltas of each window
__global__ void calcDeltaSums(unsigned char* imageDeltaArray, unsigned int* summedUpDeltaArray, int windowDimY, int windowDimX, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;
	int cz = threadIdx.z;
	int windowIndexX = cx / windowDimX;
	int windowIndexY = cy / windowDimY;

	// Check if the thread is inside the frame
	if (cz < 3 && cy < dimY && cx < dimX) {
		atomicAdd(&summedUpDeltaArray[(windowIndexY * windowDimY) * dimX + (windowIndexX * windowDimX)], imageDeltaArray[cz * dimY * dimX + cy * dimX + cx]);
	}
}

// Kernal that normalizes all the pixel deltas of each window
__global__ void normalizeDeltaSums(unsigned int* summedUpDeltaArray, float* normalizedDeltaArray, int* offsetArray, int windowDimY, int windowDimX, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if the thread is a window represent
	if (cy % windowDimY == 0 && cx % windowDimX == 0) {
		// Get the current window information
		int offsetX = offsetArray[cy * dimX + cx];
		int offsetY = offsetArray[dimY * dimX + cy * dimX + cx];

		// Calculate the number of pixels in the window
		int numPixels = windowDimY * windowDimX;

		// Calculate the not overlapping pixels
		int numNotOverlappingPixels = 0;
		int overlapX = 0;
		int overlapY = 0;

		// Calculate the number of not overlapping pixels
		if (cx + windowDimX + fabsf(offsetX) > dimX) {
			overlapX = fabsf(offsetX);
		}
		else if (cx - offsetX < 0) {
			overlapX = fabsf(offsetX);
		}
		else {
			overlapX = 0;
		}

		if (cy + windowDimY + fabsf(offsetY) > dimY) {
			overlapY = fabsf(offsetY);
		}
		else if (cy - offsetY < 0) {
			overlapY = fabsf(offsetY);
		}
		else {
			overlapY = 0;
		}

		numNotOverlappingPixels = overlapY * overlapX;
		numPixels -= numNotOverlappingPixels;

		// Normalize the summed up delta
		normalizedDeltaArray[cy * dimX + cx] = (float)summedUpDeltaArray[cy * dimX + cx] / numPixels;
	}
}

// Kernal that compares two arrays to find the lowest values
__global__ void compareArrays(float* normalizedDeltaArrayOld, float* normalizedDeltaArrayNew, bool* isValueDecreasedArray, int windowDimY, int windowDimX, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if the thread is a window represent
	if (cy % windowDimY == 0 && cx % windowDimX == 0) {
		// Compare the two values
		isValueDecreasedArray[cy * dimX + cx] = normalizedDeltaArrayNew[cy * dimX + cx] < normalizedDeltaArrayOld[cy * dimX + cx];
	}
}

// Kernal that adjusts the offset array based on the comparison results
__global__ void compositeOffsetArray(int* offsetArray, bool* isValueDecreasedArray, int* statusArray, int currentGlobalOffset, int windowDimY, int windowDimX, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;
	int wx = (cx / windowDimX) * windowDimX;
	int wy = (cy / windowDimY) * windowDimY;

	/*
	* Status Array Key:
	* 0: Set the initial positive x direction
	* 1: Test the positive x direction
	* 2: Continue moving in the positive x direction
	* 3: Set the initial negative x direction
	* 4: Test the negative x direction
	* 5: Continue moving in the negative x direction
	* 6: Set the initial positive y direction
	* 7: Test the positive y direction
	* 8: Continue moving in the positive y direction
	* 9: Set the initial negative y direction
	* 10: Test the negative y direction
	* 11: Continue moving in the negative y direction
	* 12: Search complete
	*/

	if (cy < dimY && cx < dimX) {
		int currentStatus = statusArray[cy * dimX + cx];

		switch (currentStatus) {
			/*
			* X - DIRECTION
			*/
		case 0:
			// Set the initial positive x direction
			statusArray[cy * dimX + cx] = 1;
			offsetArray[cy * dimX + cx] += currentGlobalOffset;
			break;
		case 1:
			// Test the positive x direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				statusArray[cy * dimX + cx] = 2;
				offsetArray[cy * dimX + cx] += currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 3;
				offsetArray[cy * dimX + cx] -= currentGlobalOffset;
			}
			break;
		case 2:
			// Continue moving in the positive x direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				offsetArray[cy * dimX + cx] += currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 6;
				offsetArray[cy * dimX + cx] -= currentGlobalOffset;
			}
			break;
		case 3:
			// Set the initial negative x direction
			statusArray[cy * dimX + cx] = 4;
			offsetArray[cy * dimX + cx] -= currentGlobalOffset;
			break;
		case 4:
			// Test the negative x direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				statusArray[cy * dimX + cx] = 5;
				offsetArray[cy * dimX + cx] -= currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 6;
				offsetArray[cy * dimX + cx] += currentGlobalOffset;
			}
			break;
		case 5:
			// Continue moving in the negative x direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				offsetArray[cy * dimX + cx] -= currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 6;
				offsetArray[cy * dimX + cx] += currentGlobalOffset;
			}
			break;

			/*
			* Y - DIRECTION
			*/
		case 6:
			// Set the initial positive y direction
			statusArray[cy * dimX + cx] = 7;
			offsetArray[dimY * dimX + cy * dimX + cx] += currentGlobalOffset;
			break;
		case 7:
			// Test the positive y direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				statusArray[cy * dimX + cx] = 8;
				offsetArray[dimY * dimX + cy * dimX + cx] += currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 9;
				offsetArray[dimY * dimX + cy * dimX + cx] -= currentGlobalOffset;
			}
			break;
		case 8:
			// Continue moving in the positive y direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				offsetArray[dimY * dimX + cy * dimX + cx] += currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 12;
				offsetArray[dimY * dimX + cy * dimX + cx] -= currentGlobalOffset;
			}
			break;
		case 9:
			// Set the initial negative y direction
			statusArray[cy * dimX + cx] = 10;
			offsetArray[dimY * dimX + cy * dimX + cx] -= currentGlobalOffset;
			break;
		case 10:
			// Test the negative y direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				statusArray[cy * dimX + cx] = 11;
				offsetArray[dimY * dimX + cy * dimX + cx] -= currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 12;
				offsetArray[dimY * dimX + cy * dimX + cx] += currentGlobalOffset;
			}
			break;
		case 11:
			// Continue moving in the negative y direction
			if (isValueDecreasedArray[wy * dimX + wx]) {
				offsetArray[dimY * dimX + cy * dimX + cx] -= currentGlobalOffset;
			}
			else {
				statusArray[cy * dimX + cx] = 12;
				offsetArray[dimY * dimX + cy * dimX + cx] += currentGlobalOffset;
			}
			break;
		case 12:
			// Search is complete
			break;
		default:
			break;
		}
	}
	return;
}

// Kernal that warps frame1 according to the offset array
__global__ void warpFrameKernal(unsigned char* frame1, int* offsetArray, int* hitCount, int* ones, unsigned char* warpedFrame, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;
	int cz = threadIdx.z;

	// Get the current offsets to use
	int offsetX = offsetArray[cy * dimX + cx];
	int offsetY = offsetArray[dimY * dimX + cy * dimX + cx];

	// Check if the thread is inside the frame (without offsets)
	if (cz < 3 && cy < dimY && cx < dimX) {
		// Check if the current pixel is inside of the frame
		if ((cy + offsetY >= 0) && (cy + offsetY < dimY) && (cx + offsetX >= 0) && (cx + offsetX < dimX)) {
			warpedFrame[cz * dimY * dimX + cy * dimX + cx + (offsetY * dimX + offsetX)] = frame1[cz * dimY * dimX + cy * dimX + cx];
			atomicAdd(&hitCount[cz * dimY * dimX + cy * dimX + cx + (offsetY * dimX + offsetX)], ones[cz * dimY * dimX + cy * dimX + cx]);
		}
	}
}

// Kernal that removes artifacts from the warped frame
__global__ void artifactRemovalKernal(unsigned char* frame1, int* hitCount, unsigned char* warpedFrame, int dimY, int dimX) {
	// Current entry to be computed by the thread
	int cx = blockIdx.x * blockDim.x + threadIdx.x;
	int cy = blockIdx.y * blockDim.y + threadIdx.y;
	int cz = threadIdx.z;

	// Check if the thread is inside the frame (without offsets)
	if (cz < 3 && cy < dimY && cx < dimX) {
		// Check if the current pixel is inside of the frame
		if (hitCount[cz * dimY * dimX + cy * dimX + cx] != 1) {
			warpedFrame[cz * dimY * dimX + cy * dimX + cx] = frame1[cz * dimY * dimX + cy * dimX + cx];
		}
	}
}

/*
* Calculates the optical flow between frame1 and frame2
*
* @param frame1: The frame to calculate the flow from
* @param frame2: The frame to calculate the flow to
*
* @return: The flow array containing the relative vectors
*/
GPUArray<int> calculateOpticalFlow(GPUArray<unsigned char>& frame1, GPUArray<unsigned char>& frame2) {
	if (DEBUG_MODE) {
		// Check if the dimensions match
		if (frame1.dimZ != 3 || frame2.dimZ != 3 || frame1.dimY != frame2.dimY || frame1.dimX != frame2.dimX) {
			fprintf(stderr, "ERROR: Frame dimensions do not match!\n");
			exit(-1);
		}

		// Check if the frames are on the GPU
		if (!frame1.isOnGPU) {
			frame1.toGPU();
		}
		if (!frame2.isOnGPU) {
			frame2.toGPU();
		}
	}

	// Calculate the number of cuda blocks needed
	int NUM_BLOCKS_X = fmaxf(ceilf(frame1.dimX / NUM_THREADS), 1);
	int NUM_BLOCKS_Y = fmaxf(ceilf(frame1.dimY / NUM_THREADS), 1);

	// Calculate the number of cuda threads needed
	dim3 grid(NUM_BLOCKS_X, NUM_BLOCKS_Y, 1);
	dim3 threads3(NUM_THREADS, NUM_THREADS, 3);
	dim3 threads2(NUM_THREADS, NUM_THREADS, 2);
	dim3 threads1(NUM_THREADS, NUM_THREADS, 1);

	// Initialize result arrays
	GPUArray<unsigned char> imageDeltaArray(frame1.shape, 0); // Array containing the absolute difference between the two frames
	GPUArray<int> offsetArray({ 2, frame1.dimY, frame1.dimX }, 0); // Array containing x,y offsets for each pixel of frame1
	GPUArray<int> statusArray({ frame1.dimY, frame1.dimX }, 0); // Array containing the calculation status of each pixel of frame1
	GPUArray<unsigned int> summedUpDeltaArray({ frame1.dimY, frame1.dimX }, 0); // Array containing the summed up delta values of each window
	GPUArray<float> normalizedDeltaArrayA({ frame1.dimY, frame1.dimX }, 0); // Array containing the normalized delta values of each window
	GPUArray<float> normalizedDeltaArrayB({ frame1.dimY, frame1.dimX }, 0); // Array containing the normalized delta values of each window
	GPUArray<bool> isValueDecreasedArray({ frame1.dimY, frame1.dimX }, 0); // Array containing the comparison results of the two normalized delta arrays (true if the new value decreased)
	int windowDimX = frame1.dimX; // Initial window size
	int windowDimY = frame1.dimY; // Initial window size
	int currentGlobalOffset = fmax(frame1.dimX / MAX_OFFSET_DIVIDER, 1); // Initial global offset
	int numIterations = ceil(log2f(frame1.dimX)); // Number of iterations needed to get to the smallest window size

	auto start = std::chrono::high_resolution_clock::now();

	// We calculate the ideal offset array for each window size (entire frame, ..., individual pixels)
	for (int iter = 0; iter < numIterations; iter++) {
		// Each step we adjust the offset array to find the ideal offset
		for (int step = 0; step < NUM_STEPS; step++) {
			// Calculate the image deltas with the current offset array
			calcImageDelta << <grid, threads3 >> > (frame1.arrayPtrGPU, frame2.arrayPtrGPU, imageDeltaArray.arrayPtrGPU, offsetArray.arrayPtrGPU, frame1.dimY, frame1.dimX);

			// Sum up the deltas of each window
			calcDeltaSums << <grid, threads3 >> > (imageDeltaArray.arrayPtrGPU, summedUpDeltaArray.arrayPtrGPU, windowDimY, windowDimX, frame1.dimY, frame1.dimX);

			// Switch between the two normalized delta arrays to avoid copying
			if (step % 2 == 0) {
				// Normalize the summed up delta array
				normalizeDeltaSums << <grid, threads1 >> > (summedUpDeltaArray.arrayPtrGPU, normalizedDeltaArrayB.arrayPtrGPU, offsetArray.arrayPtrGPU, windowDimY, windowDimX, frame1.dimY, frame1.dimX);

				if (DEBUG_MODE) {
					printf("Offset Xe:\n");
					offsetArray.print<int>(0, 1);
					offsetArray.toGPU();
					printf("Offset Ye:\n");
					offsetArray.print<int>(frame1.dimY * frame1.dimX, 1);
					offsetArray.toGPU();
					printf("Normalized Delta:\n");
					normalizedDeltaArrayB.print<float>(0, 1);
					normalizedDeltaArrayB.toGPU();
					printf("\n");
				}

				// Check if the new normalized delta array is better than the old one
				compareArrays << <grid, threads1 >> > (normalizedDeltaArrayA.arrayPtrGPU, normalizedDeltaArrayB.arrayPtrGPU, isValueDecreasedArray.arrayPtrGPU, windowDimY, windowDimX, frame1.dimY, frame1.dimX);
			}
			else {
				// Normalize the summed up delta array
				normalizeDeltaSums << <grid, threads1 >> > (summedUpDeltaArray.arrayPtrGPU, normalizedDeltaArrayA.arrayPtrGPU, offsetArray.arrayPtrGPU, windowDimY, windowDimX, frame1.dimY, frame1.dimX);

				if (DEBUG_MODE) {
					printf("Offset Xu:\n");
					offsetArray.print<int>(0, 1);
					offsetArray.toGPU();
					printf("Offset Yu:\n");
					offsetArray.print<int>(frame1.dimY * frame1.dimX, 1);
					offsetArray.toGPU();
					printf("Normalized Delta:\n");
					normalizedDeltaArrayA.print<float>(0, 1);
					normalizedDeltaArrayA.toGPU();
					printf("\n");
				}

				// Check if the new normalized delta array is better than the old one
				compareArrays << <grid, threads1 >> > (normalizedDeltaArrayB.arrayPtrGPU, normalizedDeltaArrayA.arrayPtrGPU, isValueDecreasedArray.arrayPtrGPU, windowDimY, windowDimX, frame1.dimY, frame1.dimX);
			}

			// Adjust the offset array based on the comparison results
			compositeOffsetArray << <grid, threads1 >> > (offsetArray.arrayPtrGPU, isValueDecreasedArray.arrayPtrGPU, statusArray.arrayPtrGPU, currentGlobalOffset, windowDimY, windowDimX, frame1.dimY, frame1.dimX);

			// Wait for all threads to finish
			hipDeviceSynchronize();

			// Reset the summed up delta array
			summedUpDeltaArray.fill(0);
		}
		// Adjust window size
		windowDimX = fmax(windowDimX / 2, 1);
		windowDimY = fmax(windowDimY / 2, 1);

		// Adjust global offset
		currentGlobalOffset = fmax(currentGlobalOffset / 2, 1);

		// Reset the status array
		statusArray.fill(0);
	}

	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration<double, std::milli>(stop - start).count();
	std::cout << "\nOpt. Flow Calc Time: " << std::fixed << std::setprecision(4) << duration << " milliseconds" << std::endl;

	// Check for CUDA errors
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}

	// Free memory
	imageDeltaArray.del();
	statusArray.del();
	summedUpDeltaArray.del();
	normalizedDeltaArrayA.del();
	normalizedDeltaArrayB.del();
	isValueDecreasedArray.del();

	// Return result array
	return offsetArray;
}

/*
* Warps frame1 according to the offset array
*
* @param frame1: The frame to warp
* @param offsetArray: The array containing the offsets
*
* @return: The warped frame
*/
GPUArray<unsigned char> warpFrame(GPUArray<unsigned char>& frame1, GPUArray<int>& offsetArray) {
	if (DEBUG_MODE) {
		// Check if the dimensions match
		if (frame1.dimZ != 3 || offsetArray.dimZ != 3 || frame1.dimY != offsetArray.dimY || frame1.dimX != offsetArray.dimX) {
			fprintf(stderr, "ERROR: Frame and offset array dimensions do not match!\n");
			exit(-1);
		}

		// Check if the arrays are on the GPU
		if (!frame1.isOnGPU) {
			frame1.toGPU();
		}
		if (!offsetArray.isOnGPU) {
			offsetArray.toGPU();
		}
	}

	// Calculate the number of cuda blocks needed
	int NUM_BLOCKS_X = fmaxf(ceilf(frame1.dimX / NUM_THREADS), 1);
	int NUM_BLOCKS_Y = fmaxf(ceilf(frame1.dimY / NUM_THREADS), 1);

	// Calculate the number of cuda threads needed
	dim3 grid(NUM_BLOCKS_X, NUM_BLOCKS_Y, 1);
	dim3 threads3(NUM_THREADS, NUM_THREADS, 3);

	// Initialize result arrays
	GPUArray<unsigned char> warpedFrame(frame1.shape, 0); // Array containing the warped frame
	GPUArray<int> hitCount(frame1.shape, 0); // Array containing the number of times a pixel was hit
	GPUArray<int> ones(frame1.shape, 1); // Array containing only ones for atomic add

	auto start = std::chrono::high_resolution_clock::now();

	// Warp the frame
	warpFrameKernal << <grid, threads3 >> > (frame1.arrayPtrGPU, offsetArray.arrayPtrGPU, hitCount.arrayPtrGPU, ones.arrayPtrGPU, warpedFrame.arrayPtrGPU, frame1.dimY, frame1.dimX);

	// Wait for all threads to finish
	hipDeviceSynchronize();

	// Remove artifacts
	artifactRemovalKernal << <grid, threads3 >> > (frame1.arrayPtrGPU, hitCount.arrayPtrGPU, warpedFrame.arrayPtrGPU, frame1.dimY, frame1.dimX);

	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration<double, std::milli>(stop - start).count();
	std::cout << "\nWarp Calc Time: " << std::fixed << std::setprecision(4) << duration << " milliseconds" << std::endl;

	// Check for CUDA errors
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}

	// Return result array
	return warpedFrame;
}