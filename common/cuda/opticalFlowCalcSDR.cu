#include "hip/hip_runtime.h"
#include <amvideo.h>
#include <iomanip>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include "opticalFlowCalcSDR.cuh"

// Debug message function
void CudaDebugMessage(const std::string& message, const bool showLog) {
	if (showLog) {
		const std::string m_debugMessage = message + "\n";
		OutputDebugStringA(m_debugMessage.c_str());
	}
}

// Kernel that converts an NV12 array to a P010 array
__global__ void convertNV12toP010KernelSDR(const unsigned char* nv12Array, unsigned short* p010Array, const unsigned int dimY,
										   const unsigned int dimX, const int scaledDimY, const int scaledDimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Check if the current thread is inside the Y-Channel or the U/V-Channel
	if ((cz == 0 && cy < scaledDimY && cx < scaledDimX) || (cz == 1 && cy < (scaledDimY / 2) && cx < scaledDimX)) {
		p010Array[cz * dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(nv12Array[cz * dimY * dimX + cy * dimX + cx]) << 8;
	}
}

// Kernel that blurs a frame
__global__ void blurFrameKernelSDR(const unsigned char* frameArray, unsigned char* blurredFrameArray, 
								const unsigned char kernelSize, const unsigned char chacheSize, const unsigned char boundsOffset, 
								const unsigned char avgEntriesPerThread, const unsigned short remainder, const char lumStart,
								const unsigned char lumEnd, const unsigned short lumPixelCount, const char chromStart, 
								const unsigned char chromEnd, const unsigned short chromPixelCount, const unsigned short dimY, const unsigned short dimX) {
	// Shared memory for the frame to prevent multiple global memory accesses
	extern __shared__ unsigned char sharedFrameArray[];

	// Current entry to be computed by the thread
	const unsigned short cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned short cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned char cz = blockIdx.z;

	// Check if the current thread is supposed to perform calculations
	if (cz == 1 && (cy >= dimY / 2 || cx >= dimX)) {
		return;
	}

	const unsigned short trX = blockIdx.x * blockDim.x;
	const unsigned short trY = blockIdx.y * blockDim.y;
	unsigned char offsetX;
	unsigned char offsetY;

    // Calculate the number of entries to fill for this thread
    const unsigned short threadIndex = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned char entriesToFill = avgEntriesPerThread + (threadIndex < remainder ? 1 : 0);

    // Calculate the starting index for this thread
    unsigned short startIndex = 0;
    for (unsigned short i = 0; i < threadIndex; ++i) {
        startIndex += avgEntriesPerThread + (i < remainder ? 1 : 0);
    }

    // Fill the shared memory for this thread
    for (unsigned short i = 0; i < entriesToFill; ++i) {
		offsetX = (startIndex + i) % chacheSize;
		offsetY = (startIndex + i) / chacheSize;
		if ((trY - boundsOffset + offsetY) < dimY && (trX - boundsOffset + offsetX) < dimX) {
			sharedFrameArray[startIndex + i] = frameArray[cz * dimY * dimX + (trY - boundsOffset + offsetY) * dimX + (trX - boundsOffset + offsetX)];
		} else {
			sharedFrameArray[startIndex + i] = 0;
		}
	}

    // Ensure all threads have finished loading before continuing
    __syncthreads();

	// Calculate the x and y boundaries of the kernel
	unsigned int blurredPixel = 0;

	// Collect the sum of the surrounding pixels
	// Y-Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		for (char y = lumStart; y < lumEnd; y++) {
			for (char x = lumStart; x < lumEnd; x++) {
				if ((cy + y) < dimY && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset + y) * chacheSize + threadIdx.x + boundsOffset + x];
				} else {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset) * chacheSize + threadIdx.x + boundsOffset];
				}
			}
		}
		blurredPixel /= lumPixelCount;
		blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
	// U/V-Channel
	} else if (cz == 1 && cy < dimY / 2 && cx < dimX) {
		for (char y = chromStart; y < chromEnd; y++) {
			for (char x = chromStart; x < chromEnd; x++) {
				if ((cy + y) < dimY / 2 && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset + y) * chacheSize + threadIdx.x + boundsOffset + x * 2];
				} else {
					blurredPixel += sharedFrameArray[(threadIdx.y + boundsOffset) * chacheSize + threadIdx.x + boundsOffset];
				}
			}
		}
		blurredPixel /= chromPixelCount;
		blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
	}
}

// Kernel that warps a frame according to the offset array
__global__ void warpFrameKernelForOutputSDR(const unsigned char* frame1, const int* offsetArray, int* hitCount, int* ones,
									        unsigned short* warpedFrame, const float frameScalar, const int lowDimY, const int lowDimX,
											const int dimY, const int dimX, const double resolutionDivider,
											const int directionIdxOffset, const int scaledDimX, const int channelIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	const int scaledCx = static_cast<int>(cx * resolutionDivider); // The X-Index of the current thread in the offset array
	const int scaledCy = static_cast<int>(cy * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Y-Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		// Get the current offsets to use
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY && newCx >= 0 && newCx < dimX) {
			warpedFrame[newCy * scaledDimX + newCx] = static_cast<unsigned short>(frame1[cy * dimX + cx]) << 8;
			atomicAdd(&hitCount[newCy * dimX + newCx], ones[cy * dimX + cx]);
		}

	// U/V-Channel
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + 2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar / 2.0);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY / 2 && newCx >= 0 && newCx < dimX) {
			// U-Channel
			if ((cx & 1) == 0) {
				warpedFrame[channelIdxOffset + newCy * scaledDimX + (newCx / 2) * 2] = static_cast<unsigned short>(frame1[dimY * dimX + cy * dimX + cx]) << 8;

			// V-Channel
			} else {
				warpedFrame[channelIdxOffset + newCy * scaledDimX + (newCx / 2) * 2 + 1] = static_cast<unsigned short>(frame1[dimY * dimX + cy * dimX + cx]) << 8;
			}
		}
	}
}

// Kernel that warps a frame according to the offset array
__global__ void warpFrameKernelForBlendingSDR(const unsigned char* frame1, const int* offsetArray, int* hitCount, int* ones,
										      unsigned char* warpedFrame, const float frameScalar, const int lowDimY, const int lowDimX,
											  const int dimY, const int dimX, const double resolutionDivider,
											  const int directionIdxOffset, const int channelIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	const int scaledCx = static_cast<int>(cx * resolutionDivider); // The X-Index of the current thread in the offset array
	const int scaledCy = static_cast<int>(cy * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		// Get the current offsets to use
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY && newCx >= 0 && newCx < dimX) {
			warpedFrame[newCy * dimX + newCx] = frame1[cy * dimX + cx];
			atomicAdd(&hitCount[newCy * dimX + newCx], ones[cy * dimX + cx]);
		}

	// U/V-Channel
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + 2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar / 2.0);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY / 2 && newCx >= 0 && newCx < dimX) {
			// U-Channel
			if ((cx & 1) == 0) {
				warpedFrame[channelIdxOffset + newCy * dimX + (newCx / 2) * 2] = frame1[dimY * dimX + cy * dimX + cx];

			// V-Channel
			} else {
				warpedFrame[channelIdxOffset + newCy * dimX + (newCx / 2) * 2 + 1] = frame1[dimY * dimX + cy * dimX + cx];
			}
		}
	}
}

// Kernel that removes artifacts from the warped frame
__global__ void artifactRemovalKernelForOutputSDR(const unsigned char* frame1, const int* hitCount, unsigned short* warpedFrame,
										     	  const unsigned int dimY, const unsigned int dimX, const int scaledDimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const unsigned int threadIndex2D = cy * dimX + cx; // Standard thread index without Z-Dim

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		if (hitCount[threadIndex2D] != 1) {
			warpedFrame[cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[threadIndex2D]) << 8;
		}

	// U/V Channels
	} else if (cz == 1 && cy < (dimY >> 1) && cx < dimX) {
		if (hitCount[threadIndex2D] != 1) {
			warpedFrame[dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(frame1[dimY * dimX + threadIndex2D]) << 8;
		}
	}
}

// Kernel that blends frame1 to frame2
__global__ void blendFrameKernelSDR(const unsigned char* frame1, const unsigned char* frame2, unsigned short* blendedFrame,
                                    const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                    const unsigned int dimX, const int scaledDimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		blendedFrame[cy * scaledDimX + cx] = 
			static_cast<unsigned short>(
				static_cast<float>(frame1[cy * dimX + cx]) * frame1Scalar + 
				static_cast<float>(frame2[cy * dimX + cx]) * frame2Scalar
			) << 8;
	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		blendedFrame[dimY * scaledDimX + cy * scaledDimX + cx] = 
			static_cast<unsigned short>(
				static_cast<float>(frame1[dimY * dimX + cy * dimX + cx]) * frame1Scalar + 
				static_cast<float>(frame2[dimY * dimX + cy * dimX + cx]) * frame2Scalar
			) << 8;
	}
}

// Kernel that creates an HSV flow image from the offset array
__global__ void convertFlowToHSVKernelSDR(const int* flowArray, unsigned short* p010Array, const unsigned char* frame1,
                                          const double blendScalar, const int lowDimX, const unsigned int dimY, const unsigned int dimX, 
										  const double resolutionDivider, const int directionIdxOffset, const int scaledDimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	const unsigned int scaledCx = static_cast<unsigned int>(cx * resolutionDivider); // The X-Index of the current thread in the offset array
	const unsigned int scaledCy = static_cast<unsigned int>(cy * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Get the current flow values
	double x;
	double y;
	if (cz == 0) {
		x = flowArray[scaledCy * lowDimX + scaledCx];
		y = flowArray[directionIdxOffset + scaledCy * lowDimX + scaledCx];
	} else {
		x = flowArray[scaledCy * 2 * lowDimX + scaledCx];
		y = flowArray[directionIdxOffset + scaledCy * 2 * lowDimX + scaledCx];
	}

	// RGB struct
	struct RGB {
		int r, g, b;
	};

	// Calculate the angle in radians
	const double angle_rad = std::atan2(y, x);

	// Convert radians to degrees
	double angle_deg = angle_rad * (180.0 / 3.14159265359);

	// Ensure the angle is positive
	if (angle_deg < 0) {
		angle_deg += 360.0;
	}

	// Normalize the angle to the range [0, 360]
	angle_deg = fmod(angle_deg, 360.0);
	if (angle_deg < 0) {
		angle_deg += 360.0;
	}

	// Map the angle to the hue value in the HSV model
	const double hue = angle_deg / 360.0;

	// Convert HSV to RGB
	const int h_i = static_cast<int>(hue * 6.0);
	const double f = hue * 6.0 - h_i;
	const double q = 1.0 - f;

	RGB rgb;
	switch (h_i % 6) {
		case 0: rgb = { static_cast<int>(255), static_cast<int>(f * 255), 0 }; break;
		case 1: rgb = { static_cast<int>(q * 255), static_cast<int>(255), 0 }; break;
		case 2: rgb = { 0, static_cast<int>(255), static_cast<int>(f * 255) }; break;
		case 3: rgb = { 0, static_cast<int>(q * 255), static_cast<int>(255) }; break;
		case 4: rgb = { static_cast<int>(f * 255), 0, static_cast<int>(255) }; break;
		case 5: rgb = { static_cast<int>(255), 0, static_cast<int>(q * 255) }; break;
		default: rgb = { 0, 0, 0 }; break;
	}

	// Prevent random colors when there is no flow
	if (fabs(x) < 1.0 && fabs(y) < 1.0) {
		rgb = { 0, 0, 0 };
	}

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		p010Array[cy * scaledDimX + cx] = 
			static_cast<unsigned short>(
				(fmax(fmin(0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b, 255.0), 0.0)) * blendScalar + 
				frame1[cy * dimX + cx] * (1.0 - blendScalar)
			) << 8;
	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		// U Channel
		if ((cx & 1) == 0) {
			p010Array[dimY * scaledDimX + cy * scaledDimX + (cx / 2) * 2] = 
				static_cast<unsigned short>(
					fmax(fmin(0.492 * (rgb.b - (0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b)) + 128, 255.0), 0.0)
				) << 8;
		// V Channel
		} else {
			p010Array[dimY * scaledDimX + cy * scaledDimX + (cx / 2) * 2 + 1] = 
				static_cast<unsigned short>(
					fmax(fmin(0.877 * (rgb.r - (0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b)) + 128, 255.0), 0.0)
				) << 8;
		}
	}
}

/*
* Initializes the SDR optical flow calculator
*
* @param dimY: The height of the frame
* @param dimX: The width of the frame
* @param dDimScalar: The scalar to scale the frame dimensions with depending on the renderer used
* @param dResolutionDivider: The scalar to scale the resolution with
*/
OpticalFlowCalcSDR::OpticalFlowCalcSDR(const unsigned int dimY, const unsigned int dimX, const double dDimScalar, const double dResolutionDivider) {
	m_dResolutionDivider = dResolutionDivider;
	m_dResolutionScalar = 1.0 / dResolutionDivider;
	m_iDimX = dimX;
	m_iDimY = dimY;
	m_iLowDimX = static_cast<unsigned int>(static_cast<double>(dimX) * m_dResolutionDivider);
	m_iLowDimY = static_cast<unsigned int>(static_cast<double>(dimY) * m_dResolutionDivider);
	m_iNumLayers = 5;
	m_dDimScalar = dDimScalar;
	m_lowGrid.x = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimX) / static_cast<double>(NUM_THREADS)), 1.0));
	m_lowGrid.y = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimY) / static_cast<double>(NUM_THREADS)), 1.0));
	m_lowGrid.z = 1;
	m_gridCID.x = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimX) / static_cast<double>(16)), 1.0));
	m_gridCID.y = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimY) / static_cast<double>(16)), 1.0));
	m_gridCID.z = m_iNumLayers;
	m_gridAOA.x = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimX) / static_cast<double>(32)), 1.0));
	m_gridAOA.y = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimY) / static_cast<double>(32)), 1.0));
	m_gridAOA.z = 1;
	m_threadsCID.x = 16;
	m_threadsCID.y = 16;
	m_threadsCID.z = 2;
	m_threadsAOA.x = 32;
	m_threadsAOA.y = 32;
	m_threadsAOA.z = 1;
	m_threads10.x = NUM_THREADS;
	m_threads10.y = NUM_THREADS;
	m_threads10.z = 10;
	m_threads5.x = NUM_THREADS;
	m_threads5.y = NUM_THREADS;
	m_threads5.z = 5;
	m_threads2.x = NUM_THREADS;
	m_threads2.y = NUM_THREADS;
	m_threads2.z = 2;
	m_threads1.x = NUM_THREADS;
	m_threads1.y = NUM_THREADS;
	m_threads1.z = 1;
	m_grid.x = static_cast<int>(fmax(ceil(dimX / static_cast<double>(NUM_THREADS)), 1.0));
	m_grid.y = static_cast<int>(fmax(ceil(dimY / static_cast<double>(NUM_THREADS)), 1.0));
	m_frame1.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_frame2.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_blurredFrame1.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_blurredFrame2.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_imageDeltaArray.init({5, 2, dimY, dimX});
	m_offsetArray12.init({2, 5, dimY, dimX});
	m_offsetArray21.init({2, dimY, dimX});
	m_blurredOffsetArray12.init({2, dimY, dimX});
	m_blurredOffsetArray21.init({2, dimY, dimX});
	m_statusArray.init({dimY, dimX});
	m_summedUpDeltaArray.init({5, dimY, dimX});
	m_normalizedDeltaArray.init({5, dimY, dimX});
	m_lowestLayerArray.init({dimY, dimX});
	m_warpedFrame12.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_warpedFrame21.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_outputFrame.init({1, dimY, dimX}, 0, static_cast<size_t>(3.0 * dimY * dimX * dDimScalar));
	m_hitCount12.init({1, dimY, dimX});
	m_hitCount21.init({1, dimY, dimX});
	m_ones.init({1, dimY, dimX}, 1);
}

/*
* Updates the frame1 array
*
* @param pInBuffer: Pointer to the input frame
*/
void OpticalFlowCalcSDR::updateFrame1(const unsigned char* pInBuffer) {
	m_frame1.fillData(pInBuffer);
	m_bBisNewest = false;
}

/*
* Updates the frame2 array
*
* @param pInBuffer: Pointer to the input frame
*/
void OpticalFlowCalcSDR::updateFrame2(const unsigned char* pInBuffer) {
	m_frame2.fillData(pInBuffer);
	m_bBisNewest = true;
}

/*
* Copies the frame in the correct format to the output buffer
*
* @param pInBuffer: Pointer to the input frame
* @param pOutBuffer: Pointer to the output frame
*/
void OpticalFlowCalcSDR::copyFrame(const unsigned char* pInBuffer, unsigned char* pOutBuffer) {
	// Set the array entries to the provided value
	m_frame1.fillData(pInBuffer);

	// Convert the NV12 frame to P010
	convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_iDimY * m_dDimScalar, m_iDimX * m_dDimScalar);

	// Download the output frame
	m_outputFrame.download(pOutBuffer);
}

/*
* Blurs a frame
*
* @param kernelSize: Size of the kernel to use for the blur
* @param directOutput: Whether to output the blurred frame directly
*/
void OpticalFlowCalcSDR::blurFrameArray(const unsigned char kernelSize, const bool directOutput) {
	const unsigned char boundsOffset = kernelSize >> 1;
	const unsigned char chacheSize = kernelSize + (boundsOffset << 1);
	const size_t sharedMemSize = chacheSize * chacheSize * sizeof(unsigned char);
	const unsigned short totalThreads = max(kernelSize * kernelSize, 1);
    const unsigned short totalEntries = chacheSize * chacheSize;
    const unsigned char avgEntriesPerThread = totalEntries / totalThreads;
	const unsigned short remainder = totalEntries % totalThreads;
	const char lumStart = -(kernelSize >> 1);
	const unsigned char lumEnd = (kernelSize >> 1);
	const char chromStart = -(kernelSize >> 2);
	const unsigned char chromEnd = (kernelSize >> 2);
	const unsigned short lumPixelCount = (lumEnd - lumStart) * (lumEnd - lumStart);
	const unsigned short chromPixelCount = (chromEnd - chromStart) * (chromEnd - chromStart);

	// Calculate the number of blocks needed
	const unsigned int NUM_BLOCKS_X = max(static_cast<int>(ceil(static_cast<float>(m_iDimX) / kernelSize)), 1);
	const unsigned int NUM_BLOCKS_Y = max(static_cast<int>(ceil(static_cast<float>(m_iDimY) / kernelSize)), 1);

	// Use dim3 structs for block and grid size
	dim3 gridBF(NUM_BLOCKS_X, NUM_BLOCKS_Y, 2);
	dim3 threadsBF(kernelSize, kernelSize, 1);

	if (!m_bBisNewest) {
		// No need to blur the frame if the kernel size is less than 4
		if (kernelSize < 4) {
			hipMemcpy(m_blurredFrame1.arrayPtrGPU, m_frame1.arrayPtrGPU, m_frame1.bytes, hipMemcpyDeviceToDevice);
		} else {
			// Launch kernel
			blurFrameKernelSDR << <gridBF, threadsBF, sharedMemSize >> > (m_frame1.arrayPtrGPU, m_blurredFrame1.arrayPtrGPU, kernelSize, chacheSize, boundsOffset, avgEntriesPerThread, remainder, lumStart, lumEnd, lumPixelCount, chromStart, chromEnd, chromPixelCount, m_iDimY, m_iDimX);
		}

		// Convert the NV12 frame to P010 if we are doing direct output
		if (directOutput) {
			convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_blurredFrame1.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_iDimY * m_dDimScalar, m_iDimX * m_dDimScalar);
		}
	} else {
		// No need to blur the frame if the kernel size is less than 4
		if (kernelSize < 4) {
			hipMemcpy(m_blurredFrame2.arrayPtrGPU, m_frame2.arrayPtrGPU, m_frame1.bytes, hipMemcpyDeviceToDevice);
		} else {
			// Launch kernel
			blurFrameKernelSDR << <gridBF, threadsBF, sharedMemSize >> > (m_frame2.arrayPtrGPU, m_blurredFrame2.arrayPtrGPU, kernelSize, chacheSize, boundsOffset, avgEntriesPerThread, remainder, lumStart, lumEnd, lumPixelCount, chromStart, chromEnd, chromPixelCount, m_iDimY, m_iDimX);
		}

		// Convert the NV12 frame to P010 if we are doing direct output
		if (directOutput) {
			convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_blurredFrame2.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_iDimY * m_dDimScalar, m_iDimX * m_dDimScalar);
		}
	}
}

/*
* Calculates the optical flow between frame1 and frame2
*
* @param iNumIterations: Number of iterations to calculate the optical flow
* @param iNumSteps: Number of steps executed to find the ideal offset (limits the maximum offset)
*/
void OpticalFlowCalcSDR::calculateOpticalFlow(unsigned int iNumIterations, unsigned int iNumSteps) {
	// Reset variables
	const unsigned int directionIdxOffset = m_iNumLayers * m_iLowDimY * m_iLowDimX; // Offset to index the Y-Offset-Layer
	const unsigned int channelIdxOffset = m_iDimY * m_iDimX; // Offset to index the color channel of the current thread
	const unsigned int layerIdxOffset = m_iLowDimY * m_iLowDimX; // Offset to index the layer of the current thread
	unsigned int iNumStepsPerIter = iNumSteps; // Number of steps executed to find the ideal offset (limits the maximum offset)

	// We set the initial window size to the next larger power of 2
	unsigned int windowDim = 1;
	unsigned int maxDim = max(m_iLowDimX, m_iLowDimY);
    if (maxDim && !(maxDim & (maxDim - 1))) {
		windowDim = maxDim;
	} else {
		while (maxDim & (maxDim - 1)) {
			maxDim &= (maxDim - 1);
		}
		windowDim = maxDim << 1;
	}

	if (iNumIterations == 0 || static_cast<double>(iNumIterations) > ceil(log2(windowDim))) {
		iNumIterations = static_cast<unsigned int>(ceil(log2(windowDim))) + 1;
	}

	unsigned int num_threads = min(windowDim, 16);
	size_t sharedMemSize = num_threads * num_threads * sizeof(unsigned int);

	// Calculate the number of blocks needed
	unsigned int NUM_BLOCKS_X = max(static_cast<int>(ceil(static_cast<float>(m_imageDeltaArray.dimX) / num_threads)), 1);
	unsigned int NUM_BLOCKS_Y = max(static_cast<int>(ceil(static_cast<float>(m_imageDeltaArray.dimY) / num_threads)), 1);

	// Use dim3 structs for block and grid size
	dim3 gridCDS(NUM_BLOCKS_X, NUM_BLOCKS_Y, 5);
	dim3 threadsCDS(num_threads, num_threads, 1);

	// Set the starting offset for the current window size
	setInitialOffset << <m_lowGrid, m_threads5 >> > (m_offsetArray12.arrayPtrGPU, m_iNumLayers, m_iLowDimY, m_iLowDimX);

	// We calculate the ideal offset array for each window size (entire frame, ..., individual pixels)
	for (unsigned int iter = 0; iter < iNumIterations; iter++) {
		switch (iter) {
			case 0: iNumStepsPerIter = iNumSteps; break;
			case 1: iNumStepsPerIter = iNumSteps; break;
			case 2: iNumStepsPerIter = iNumSteps; break;
			case 3: iNumStepsPerIter = iNumSteps / 2; break;
			case 4: iNumStepsPerIter = iNumSteps / 2; break;
			case 5: iNumStepsPerIter = iNumSteps / 2; break;
			case 6: iNumStepsPerIter = iNumSteps / 3; break;
			case 7: iNumStepsPerIter = iNumSteps / 3; break;
			case 8: iNumStepsPerIter = iNumSteps / 6; break;
			default: iNumStepsPerIter = 1; break;
		}
		// Each step we adjust the offset array to find the ideal offset
		for (unsigned int step = 0; step < iNumStepsPerIter; step++) {
			// Reset the summed up delta array
			m_summedUpDeltaArray.zero();

			// 1. Calculate the image deltas with the current offset array
			if (m_bBisNewest) {
				calcImageDelta << <m_gridCID, m_threadsCID >> > (m_blurredFrame1.arrayPtrGPU, m_blurredFrame2.arrayPtrGPU,
															   m_imageDeltaArray.arrayPtrGPU, m_offsetArray12.arrayPtrGPU,
															   m_iLowDimY, m_iLowDimX, m_iDimY, m_iDimX,
															   static_cast<float>(m_dResolutionScalar), directionIdxOffset, channelIdxOffset);
			} else {
				calcImageDelta << <m_gridCID, m_threadsCID >> > (m_blurredFrame2.arrayPtrGPU, m_blurredFrame1.arrayPtrGPU,
															   m_imageDeltaArray.arrayPtrGPU, m_offsetArray12.arrayPtrGPU,
															   m_iLowDimY, m_iLowDimX, m_iDimY, m_iDimX,
															   static_cast<float>(m_dResolutionScalar), directionIdxOffset, channelIdxOffset);
			}

			// 2. Sum up the deltas of each window
			if (windowDim >= 8) {
				calcDeltaSums8x8 << <gridCDS, threadsCDS, sharedMemSize>> > (m_imageDeltaArray.arrayPtrGPU, m_summedUpDeltaArray.arrayPtrGPU, 
																	2 * m_iLowDimY * m_iLowDimX, m_iLowDimY * m_iLowDimX, 
																	m_iLowDimY, m_iLowDimX, windowDim);
			} else if (windowDim == 4) {
				calcDeltaSums4x4 << <gridCDS, threadsCDS, sharedMemSize>> > (m_imageDeltaArray.arrayPtrGPU, m_summedUpDeltaArray.arrayPtrGPU, 
																	2 * m_iLowDimY * m_iLowDimX, m_iLowDimY * m_iLowDimX, 
																	m_iLowDimY, m_iLowDimX, windowDim);
			} else if (windowDim == 2) {
				calcDeltaSums2x2 << <gridCDS, threadsCDS, sharedMemSize>> > (m_imageDeltaArray.arrayPtrGPU, m_summedUpDeltaArray.arrayPtrGPU, 
																	2 * m_iLowDimY * m_iLowDimX, m_iLowDimY * m_iLowDimX, 
																	m_iLowDimY, m_iLowDimX, windowDim);
			} else if (windowDim == 1) {
				calcDeltaSums1x1 << <m_lowGrid, m_threads5, sharedMemSize>> > (m_imageDeltaArray.arrayPtrGPU, m_summedUpDeltaArray.arrayPtrGPU, 
																	2 * m_iLowDimY * m_iLowDimX, m_iLowDimY * m_iLowDimX, 
																	m_iLowDimY, m_iLowDimX, windowDim);
			}

			// 3. Normalize the summed up delta array and find the best layer
			normalizeDeltaSums << <m_lowGrid, m_threads5 >> > (m_summedUpDeltaArray.arrayPtrGPU, m_lowestLayerArray.arrayPtrGPU,
															   m_offsetArray12.arrayPtrGPU, windowDim, windowDim * windowDim,
															   directionIdxOffset, layerIdxOffset, m_iNumLayers, m_iLowDimY, m_iLowDimX);

			// 4. Adjust the offset array based on the comparison results
			adjustOffsetArray << <m_gridAOA, m_threadsAOA >> > (m_offsetArray12.arrayPtrGPU, m_lowestLayerArray.arrayPtrGPU,
															  m_statusArray.arrayPtrGPU, windowDim, directionIdxOffset, layerIdxOffset,
															  m_iNumLayers, m_iLowDimY, m_iLowDimX, step == iNumStepsPerIter - 1);
		}

		// 5. Adjust variables for the next iteration
		windowDim = max(windowDim >> 1, 1);
		num_threads = max(min(windowDim, 16), 8);
		sharedMemSize = num_threads * num_threads * sizeof(unsigned int);
		NUM_BLOCKS_X = max(static_cast<int>(ceil(static_cast<float>(m_imageDeltaArray.dimX) / num_threads)), 1);
		NUM_BLOCKS_Y = max(static_cast<int>(ceil(static_cast<float>(m_imageDeltaArray.dimY) / num_threads)), 1);
		gridCDS.x = NUM_BLOCKS_X;
		gridCDS.y = NUM_BLOCKS_Y;
		threadsCDS.x = num_threads;
		threadsCDS.y = num_threads;

		// Reset the status array
		m_statusArray.zero();
	}
}

/*
* Warps the frames according to the calculated optical flow
*
* @param fScalar: The scalar to blend the frames with
* @param bOutput12: Whether to output the warped frame 12 or 21
*/
void OpticalFlowCalcSDR::warpFramesForOutput(float fScalar, const bool bOutput12) {
	// Calculate the blend scalar
	const float frameScalar12 = fScalar;
	const float frameScalar21 = static_cast<float>(1.0) - fScalar;

	// Calculate variables so the threds don't have to do it
	const int directionIdxOffset = m_iDimY * m_dResolutionDivider * m_iDimX * m_dResolutionDivider;
	const int scaledDimX = static_cast<unsigned int>(m_iDimX * m_dDimScalar);
	const int channelIdxOffset = static_cast<unsigned int>(m_iDimY * m_iDimX * m_dDimScalar);

	// Reset the hit count array
	if (bOutput12) {
		m_hitCount12.zero();
	} else {
		m_hitCount21.zero();
	}

	// Launch kernels
	if (m_bBisNewest) {
		// Frame 1 to Frame 2
		if (bOutput12) {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
														          m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_outputFrame.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, scaledDimX);
		// Frame 2 to Frame 1
		} else {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_outputFrame.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, scaledDimX);
		}
	} else {
		// Frame 1 to Frame 2
		if (bOutput12) {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
																  m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_outputFrame.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																		m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, scaledDimX);
		// Frame 2 to Frame 1
		} else {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_outputFrame.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, scaledDimX);
		}
	}
}

/*
* Warps the frames according to the calculated optical flow
*
* @param fScalar: The scalar to blend the frames with
*/
void OpticalFlowCalcSDR::warpFramesForBlending(float fScalar) {
	// Calculate the blend scalar
	const float frameScalar12 = fScalar;
	const float frameScalar21 = static_cast<float>(1.0) - fScalar;

	// Calculate variables so the threds don't have to do it
	const int directionIdxOffset = m_iDimY * m_dResolutionDivider * m_iDimX * m_dResolutionDivider;
	const int channelIdxOffset = static_cast<unsigned int>(m_iDimY * m_iDimX);

	// Reset the hit count array
	m_hitCount12.zero();
	m_hitCount21.zero();

	// Create CUDA streams
	hipStream_t warpStream1, warpStream2;
	hipStreamCreate(&warpStream1);
	hipStreamCreate(&warpStream2);

	// Launch kernels
	if (m_bBisNewest) {
		// Frame 1 to Frame 2
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
														          m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_warpedFrame12.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, channelIdxOffset);
		artifactRemovalKernelForBlending << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame1.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																				      m_warpedFrame12.arrayPtrGPU, m_iDimY, m_iDimX);

		// Frame 2 to Frame 1
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_warpedFrame21.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, channelIdxOffset);
		artifactRemovalKernelForBlending << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame2.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																					  m_warpedFrame21.arrayPtrGPU, m_iDimY, m_iDimX);

	} else {
		// Frame 1 to Frame 2
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
																  m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_warpedFrame12.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, channelIdxOffset);
		artifactRemovalKernelForBlending << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame2.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																					  m_warpedFrame12.arrayPtrGPU, m_iDimY, m_iDimX);

		// Frame 2 to Frame 1
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_warpedFrame21.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, channelIdxOffset);
		artifactRemovalKernelForBlending << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame1.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																					  m_warpedFrame21.arrayPtrGPU, m_iDimY, m_iDimX);

	}

	// Synchronize streams to ensure completion
	hipStreamSynchronize(warpStream1);
	hipStreamSynchronize(warpStream2);

	// Clean up streams
	hipStreamDestroy(warpStream1);
	hipStreamDestroy(warpStream2);
}

/*
* Blends warpedFrame1 to warpedFrame2
*
* @param dScalar: The scalar to blend the frames with
*/
void OpticalFlowCalcSDR::blendFrames(float fScalar) {
	// Calculate the blend scalar
	const float frame1Scalar = static_cast<float>(1.0) - fScalar;
	const float frame2Scalar = fScalar;
	const int scaledDimX = static_cast<unsigned int>(m_iDimX * m_dDimScalar);

	// Blend the frames
	blendFrameKernelSDR << <m_grid, m_threads2 >> >(m_warpedFrame12.arrayPtrGPU, m_warpedFrame21.arrayPtrGPU,
												 m_outputFrame.arrayPtrGPU, frame1Scalar, frame2Scalar,
	                                             m_iDimY, m_iDimX, scaledDimX);
}

/*
* Draws the flow as an RGB image
*
* @param blendScalar: The scalar that determines how much of the source frame is blended with the flow
*/
void OpticalFlowCalcSDR::drawFlowAsHSV(const double blendScalar) const {
	// Calculate variables so the threds don't have to do it
	const int directionIdxOffset = m_iDimY * m_dResolutionDivider * m_iDimX * m_dResolutionDivider;
	const int scaledDimX = static_cast<unsigned int>(m_iDimX * m_dDimScalar);

	if (m_bBisNewest) {
		convertFlowToHSVKernelSDR << <m_grid, m_threads2 >> > (m_blurredOffsetArray12.arrayPtrGPU, m_outputFrame.arrayPtrGPU,
														    m_frame2.arrayPtrGPU, blendScalar, m_iLowDimX, m_iDimY, m_iDimX, 
															m_dResolutionDivider, directionIdxOffset, scaledDimX);
	} else {
		convertFlowToHSVKernelSDR << <m_grid, m_threads2 >> > (m_blurredOffsetArray12.arrayPtrGPU, m_outputFrame.arrayPtrGPU,
														    m_frame1.arrayPtrGPU, blendScalar, m_iLowDimX, m_iDimY, m_iDimX, 
															m_dResolutionDivider, directionIdxOffset, scaledDimX);
	}
}