#include "hip/hip_runtime.h"
#include <amvideo.h>
#include <iomanip>
#include <vector>
#include <hip/hip_runtime_api.h>
#include "opticalFlowCalcSDR.cuh"

// Kernel that converts an NV12 array to a P010 array
__global__ void convertNV12toP010KernelSDR(const unsigned char* nv12Array, unsigned short* p010Array, const unsigned int dimY,
										   const unsigned int dimX, const double dimScalar) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	
	// Dimensions of the frame scaled for the renderer
	const unsigned int scaledDimX = static_cast<unsigned int>(dimX * dimScalar);
	const unsigned int scaledDimY = static_cast<unsigned int>(dimY * dimScalar);

	// Check if the current thread is inside the Y-Channel or the U/V-Channel
	if ((cz == 0 && cy < scaledDimY && cx < scaledDimX) || (cz == 1 && cy < (scaledDimY / 2) && cx < scaledDimX)) {
		p010Array[cz * dimY * scaledDimX + cy * scaledDimX + cx] = static_cast<unsigned short>(nv12Array[cz * dimY * dimX + cy * dimX + cx]) << 8;
	}
}

// Kernel that blurs a frame
__global__ void blurFrameKernel(const unsigned char* frameArray, unsigned char* blurredFrameArray, const int kernelSize, const int dimY,
						        const int dimX) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	if (kernelSize > 3) {
		// Calculate the x and y boundaries of the kernel
		int start = -(kernelSize / 2);
		int end = (kernelSize / 2);
		unsigned int blurredPixel = 0;

		// Collect the sum of the surrounding pixels
		// Y-Channel
		if (cz == 0 && cy < dimY && cx < dimX) {
			for (int y = start; y < end; y++) {
				for (int x = start; x < end; x++) {
					if ((cy + y) < dimY && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
						blurredPixel += frameArray[cz * dimY * dimX + (cy + y) * dimX + cx + x];
					} else {
						blurredPixel += frameArray[cz * dimY * dimX + cy * dimX + cx];
					}
				}
			}
			blurredPixel /= (end - start) * (end - start);
			blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
		// U/V-Channel
		} else if (cz == 1 && cy < dimY / 2 && cx < dimX) {
			start = -(kernelSize / 4);
			end = (kernelSize / 4);
			for (int y = start; y < end; y++) {
				for (int x = start; x < end; x++) {
					if ((cy + y) < dimY / 2 && (cy + y) >= 0 && (cx + x) < dimX && (cx + x) >= 0) {
						blurredPixel += frameArray[cz * dimY * dimX + (cy + y) * dimX + cx + x * 2];
					} else {
						blurredPixel += frameArray[cz * dimY * dimX + cy * dimX + cx];
					}
				}
			}
			blurredPixel /= (end - start) * (end - start);
			blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = blurredPixel;
		}
	} else {
		if ((cz == 0 && cy < dimY && cx < dimX) || (cz == 1 && cy < dimY / 2 && cx < dimX)) {
			blurredFrameArray[cz * dimY * dimX + cy * dimX + cx] = frameArray[cz * dimY * dimX + cy * dimX + cx];
		}
	}
}

// Kernel that calculates the absolute difference between two frames using the offset array
__global__ void calcImageDeltaSDR(const unsigned char* frame1, const unsigned char* frame2, unsigned char* imageDeltaArray,
							      const int* offsetArray, const int numLayers, const int lowDimY, const int lowDimX, 
								  const int dimY, const int dimX, const double resolutionScalar, const int directionIdxOffset, 
								  const int channelIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;
	const int channel = cz % 2; // YUV channel of the current thread

	// Is the current thread supposed to perform calculations
	if (cz < numLayers * 2 && cy < lowDimY && cx < lowDimX) {
		const int layer = cz / 2; // Layer of the current thread
		const int layerOffset = layer * lowDimY * lowDimX; // Offset to index the layer of the current thread
		const int scaledCx = static_cast<int>(cx * resolutionScalar); // The X-Index of the current thread in the input frames
		const int scaledCy = static_cast<int>(cy * resolutionScalar); // The Y-Index of the current thread in the input frames
		const int evenCx = (cx / 2) * 2; // The X-Index of the current thread rounded to be even

		const int threadIndex2D = cy * lowDimX + cx; // Standard thread index without Z-Dim
		const int threadIndex3D = cz * lowDimY * lowDimX + threadIndex2D; // Standard thread index

		// Y-Channel
		if (channel == 0) {
			const int offsetX = -offsetArray[layerOffset + threadIndex2D];
			const int offsetY = -offsetArray[directionIdxOffset + layerOffset + threadIndex2D];
			const int newCx = scaledCx + offsetX;
			const int newCy = scaledCy + offsetY;

			imageDeltaArray[threadIndex3D] = (newCy < 0 || newCx < 0 || newCy >= dimY || newCx >= dimX) ? 0 : 
				abs(frame1[newCy * dimX + newCx] - frame2[scaledCy * dimX + scaledCx]);

		// U/V-Channel
		} else {
			const int offsetX = -offsetArray[layerOffset + cy * lowDimX + evenCx];
			const int offsetY = -offsetArray[directionIdxOffset + layerOffset + cy * lowDimX + evenCx];
			const int newCx = scaledCx + (offsetX / 2) * 2;
			const int newCy = scaledCy * 0.5 + offsetY * 0.5;

			imageDeltaArray[threadIndex3D] = (newCy < 0 || newCx < 0 || newCy >= dimY / 2 || newCx >= dimX) ? 0 : 
				2 * abs(frame1[channelIdxOffset + newCy * dimX + newCx] - frame2[channelIdxOffset + static_cast<int>(scaledCy * 0.5) * dimX + scaledCx]);
		}
	}
}

// Kernel that sums up all the pixel deltas of each window
__global__ void calcDeltaSumsSDR(unsigned char* imageDeltaArray, unsigned int* summedUpDeltaArray, const unsigned int windowDimY,
							     const unsigned int windowDimX, const unsigned int numLayers, const unsigned int lowDimY, const unsigned int lowDimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;
	const unsigned int windowIndexX = cx / windowDimX;
	const unsigned int windowIndexY = cy / windowDimY;
	const int layer = cz / 2; // Layer of the current thread

	// Check if the thread is inside the frame
	if (cz < numLayers * 2 && cy < lowDimY && cx < lowDimX) {
		atomicAdd(&summedUpDeltaArray[layer * lowDimY * lowDimX + (windowIndexY * windowDimY) * lowDimX + (windowIndexX * windowDimX)],
			imageDeltaArray[cz * lowDimY * lowDimX + cy * lowDimX + cx]);
	}
}

// Kernel that warps a frame according to the offset array
__global__ void warpFrameKernelForOutputSDR(const unsigned char* frame1, const int* offsetArray, int* hitCount, int* ones,
									        unsigned short* warpedFrame, const float frameScalar, const int lowDimY, const int lowDimX,
											const int dimY, const int dimX, const double resolutionDivider,
											const int directionIdxOffset, const int scaledDimX, const int channelIdxOffset) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	const int scaledCx = static_cast<int>(cx * resolutionDivider); // The X-Index of the current thread in the offset array
	const int scaledCy = static_cast<int>(cy * resolutionDivider); // The Y-Index of the current thread in the offset array

	// Y-Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		// Get the current offsets to use
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + scaledCy * lowDimX + scaledCx]) * frameScalar);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY && newCx >= 0 && newCx < dimX) {
			warpedFrame[newCy * scaledDimX + newCx] = static_cast<unsigned short>(frame1[cy * dimX + cx]) << 8;
			atomicAdd(&hitCount[newCy * dimX + newCx], ones[cy * dimX + cx]);
		}

	// U/V-Channel
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[directionIdxOffset + 2 * scaledCy * lowDimX + (scaledCx / 2) * 2]) * frameScalar / 2.0);
		const int newCx = cx + offsetX;
		const int newCy = cy + offsetY;

		// Check if the current pixel is inside the frame
		if (newCy >= 0 && newCy < dimY / 2 && newCx >= 0 && newCx < dimX) {
			// U-Channel
			if (cx % 2 == 0) {
				warpedFrame[channelIdxOffset + newCy * scaledDimX + (newCx / 2) * 2] = static_cast<unsigned short>(frame1[dimY * dimX + cy * dimX + cx]) << 8;

			// V-Channel
			} else {
				warpedFrame[channelIdxOffset + newCy * scaledDimX + (newCx / 2) * 2 + 1] = static_cast<unsigned short>(frame1[dimY * dimX + cy * dimX + cx]) << 8;
			}
		}
	}
}

// Kernel that warps a frame according to the offset array
__global__ void warpFrameKernelForBlendingSDR(const unsigned char* frame1, const int* offsetArray, int* hitCount, int* ones,
										   unsigned char* warpedFrame, const float frameScalar, const int dimY, const int dimX, 
										   const double dResolutionDivider) {
	// Current entry to be computed by the thread
	const int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const int cz = threadIdx.z;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		// Get the current offsets to use
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[static_cast<unsigned int>(cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)]) * frameScalar);
		const int offsetY = static_cast<int>(static_cast<float>(offsetArray[static_cast<unsigned int>(dimY * dResolutionDivider * dimX * dResolutionDivider) + static_cast<unsigned int>(cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)]) * frameScalar);

		// Check if the current pixel is inside the frame
		if ((cy + offsetY >= 0) && (cy + offsetY < dimY) && (cx + offsetX >= 0) && (cx + offsetX < dimX)) {
			const int newCx = fminf(fmaxf(cx + offsetX, 0), dimX - 1);
			const int newCy = fminf(fmaxf(cy + offsetY, 0), dimY - 1);
			warpedFrame[newCy * dimX + newCx] = frame1[cy * dimX + cx];
			atomicAdd(&hitCount[newCy * dimX + newCx], ones[cy * dimX + cx]);
		}

	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		const int offsetX = static_cast<int>(static_cast<float>(offsetArray[static_cast<unsigned int>(2 * cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>((cx / 2) * 2 * dResolutionDivider)]) * frameScalar);
		const int offsetY = static_cast<int>((static_cast<float>(offsetArray[static_cast<unsigned int>(dimY * dResolutionDivider * dimX * dResolutionDivider) + static_cast<unsigned int>(2 * cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>((cx / 2) * 2 * dResolutionDivider)]) * frameScalar / 2.0));

		// Check if the current pixel is inside the frame
		if ((cy + offsetY >= 0) && (cy + offsetY < dimY / 2) && (cx + offsetX >= 0) && (cx + offsetX < dimX)) {
			const int newCx = fminf(fmaxf(cx + offsetX, 0), dimX - 1);
			const int newCy = fminf(fmaxf(cy + offsetY, 0), (dimY / 2) - 1);

			// U Channel
			if (cx % 2 == 0) {
				warpedFrame[dimY * dimX + newCy * dimX + (newCx / 2) * 2] = frame1[dimY * dimX + cy * dimX + cx];

			// V Channel
			} else {
				warpedFrame[dimY * dimX + newCy * dimX + (newCx / 2) * 2 + 1] = frame1[dimY * dimX + cy * dimX + cx];
			}
		}
	}
}

// Kernel that removes artifacts from the warped frame
__global__ void artifactRemovalKernelForOutputSDR(const unsigned char* frame1, const int* hitCount, unsigned short* warpedFrame,
											   const unsigned int dimY, const unsigned int dimX, const double dDimScalar) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		if (hitCount[cy * dimX + cx] != 1) {
			warpedFrame[cy * static_cast<unsigned int>(dimX * dDimScalar) + cx] = static_cast<unsigned short>(frame1[cy * dimX + cx]) << 8;
		}

	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		if (hitCount[cy * dimX + cx] != 1) {
			warpedFrame[static_cast<unsigned int>(dimY * dimX * dDimScalar) + cy * static_cast<unsigned int>(dimX * dDimScalar) + cx] = static_cast<unsigned short>(frame1[dimY * dimX + cy * dimX + cx]) << 8;
		}
	}
}

// Kernel that removes artifacts from the warped frame
__global__ void artifactRemovalKernelForBlendingSDR(const unsigned char* frame1, const int* hitCount, unsigned char* warpedFrame,
												 const unsigned int dimY, const unsigned int dimX) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		if (hitCount[cy * dimX + cx] != 1) {
			warpedFrame[cy * dimX + cx] = frame1[cy * dimX + cx];
		}

	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		if (hitCount[cy * dimX + cx] != 1) {
			warpedFrame[dimY * dimX + cy * dimX + cx] = frame1[dimY * dimX + cy * dimX + cx];
		}
	}
}

// Kernel that blends frame1 to frame2
__global__ void blendFrameKernelSDR(const unsigned char* frame1, const unsigned char* frame2, unsigned short* blendedFrame,
                                 const float frame1Scalar, const float frame2Scalar, const unsigned int dimY,
                                 const unsigned int dimX, const double dDimScalar) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		blendedFrame[cy * static_cast<unsigned int>(dimX * dDimScalar) + cx] = static_cast<unsigned short>(static_cast<float>(frame1[cy * dimX + cx]) *
					frame1Scalar + static_cast<float>(frame2[cy * dimX + cx]) * frame2Scalar) << 8;
	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		blendedFrame[static_cast<unsigned int>(dimY * dimX * dDimScalar) + cy * static_cast<unsigned int>(dimX * dDimScalar) + cx] = static_cast<unsigned short>(static_cast<float>(frame1[dimY * dimX + cy * dimX + cx]) *
			frame1Scalar + static_cast<float>(frame2[dimY * dimX + cy * dimX + cx]) * frame2Scalar) << 8;
	}
}

// Kernel that creates an HSV flow image from the offset array
__global__ void convertFlowToHSVKernelSDR(const int* flowArray, unsigned short* p010Array, const unsigned char* frame1,
                                       const unsigned int dimY, const unsigned int dimX, const float saturation,
                                       const float value, const double dResolutionDivider, const double dDimScalar) {
	// Current entry to be computed by the thread
	const unsigned int cx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int cy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int cz = threadIdx.z;

	// Blend scalar
	constexpr float scalar = 0.5;

	// Get the current flow values
	double x;
	double y;
	if (cz == 0) {
		x = flowArray[static_cast<unsigned int>(cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)];
		y = flowArray[static_cast<unsigned int>(dimY * dResolutionDivider * dimX * dResolutionDivider) + static_cast<unsigned int>(cy * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)];
	} else {
		x = flowArray[static_cast<unsigned int>(cy * 2 * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)];
		y = flowArray[static_cast<unsigned int>(dimY * dResolutionDivider * dimX * dResolutionDivider) + static_cast<unsigned int>(cy * 2 * dResolutionDivider) * static_cast<unsigned int>(dimX * dResolutionDivider) + static_cast<unsigned int>(cx * dResolutionDivider)];
	}

	// RGB struct
	struct RGB {
		int r, g, b;
	};

	// Calculate the angle in radians
	const double angle_rad = std::atan2(y, x);

	// Convert radians to degrees
	double angle_deg = angle_rad * (180.0 / 3.14159265359);

	// Ensure the angle is positive
	if (angle_deg < 0) {
		angle_deg += 360.0;
	}

	// Normalize the angle to the range [0, 360]
	angle_deg = fmod(angle_deg, 360.0);
	if (angle_deg < 0) {
		angle_deg += 360.0;
	}

	// Map the angle to the hue value in the HSV model
	const double hue = angle_deg / 360.0;

	// Convert HSV to RGB
	const int h_i = static_cast<int>(hue * 6);
	const double f = hue * 6 - h_i;
	const double p = value * (1 - saturation);
	const double q = value * (1 - f * saturation);
	const double t = value * (1 - (1 - f) * saturation);

	RGB rgb;
	switch (h_i % 6) {
		case 0: rgb = { static_cast<int>(value * 255), static_cast<int>(t * 255), static_cast<int>(p * 255) }; break;
		case 1: rgb = { static_cast<int>(q * 255), static_cast<int>(value * 255), static_cast<int>(p * 255) }; break;
		case 2: rgb = { static_cast<int>(p * 255), static_cast<int>(value * 255), static_cast<int>(t * 255) }; break;
		case 3: rgb = { static_cast<int>(p * 255), static_cast<int>(q * 255), static_cast<int>(value * 255) }; break;
		case 4: rgb = { static_cast<int>(t * 255), static_cast<int>(p * 255), static_cast<int>(value * 255) }; break;
		case 5: rgb = { static_cast<int>(value * 255), static_cast<int>(p * 255), static_cast<int>(q * 255) }; break;
		default: rgb = { 0, 0, 0 }; break;
	}

	// Prevent random colors when there is no flow
	if (fabs(x) < 1.0 && fabs(y) < 1.0) {
		rgb = { 0, 0, 0 };
	}

	// Y Channel
	if (cz == 0 && cy < dimY && cx < dimX) {
		p010Array[cy * static_cast<unsigned int>(dimX * dDimScalar) + cx] = static_cast<unsigned short>((fmaxf(fminf(static_cast<float>(0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b), 255.0), 0.0)) * scalar + frame1[cy * dimX + cx] * (1.0 - scalar)) << 8;
	// U/V Channels
	} else if (cz == 1 && cy < (dimY / 2) && cx < dimX) {
		// U Channel
		if (cx % 2 == 0) {
			p010Array[static_cast<unsigned int>(dimY * dimX * dDimScalar) + cy * static_cast<unsigned int>(dimX * dDimScalar) + (cx / 2) * 2] = static_cast<unsigned short>(fmaxf(fminf(static_cast<float>(0.492 * (rgb.b - (0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b)) + 128), 255.0), 0.0)) << 8;
		// V Channel
		} else {
			p010Array[static_cast<unsigned int>(dimY * dimX * dDimScalar) + cy * static_cast<unsigned int>(dimX * dDimScalar) + (cx / 2) * 2 + 1] = static_cast<unsigned short>(fmaxf(fminf(static_cast<float>(0.877 * (rgb.r - (0.299 * rgb.r + 0.587 * rgb.g + 0.114 * rgb.b)) + 128), 255.0), 0.0)) << 8;
		}
	}
}

/*
* Initializes the SDR optical flow calculator
*
* @param dimY: The height of the frame
* @param dimX: The width of the frame
* @param dDimScalar: The scalar to scale the frame dimensions with depending on the renderer used
* @param dResolutionDivider: The scalar to scale the resolution with
*/
OpticalFlowCalcSDR::OpticalFlowCalcSDR(const unsigned int dimY, const unsigned int dimX, const double dDimScalar, const double dResolutionDivider) {
	m_dResolutionDivider = dResolutionDivider;
	m_dResolutionScalar = 1.0 / dResolutionDivider;
	m_iDimX = dimX;
	m_iDimY = dimY;
	m_iLowDimX = static_cast<unsigned int>(static_cast<double>(dimX) * m_dResolutionDivider);
	m_iLowDimY = static_cast<unsigned int>(static_cast<double>(dimY) * m_dResolutionDivider);
	m_iNumLayers = 5;
	m_dDimScalar = dDimScalar;
	m_lowGrid.x = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimX) / static_cast<double>(NUM_THREADS)), 1.0));
	m_lowGrid.y = static_cast<int>(fmax(ceil(static_cast<double>(m_iLowDimY) / static_cast<double>(NUM_THREADS)), 1.0));
	m_lowGrid.z = 1;
	m_threads10.x = NUM_THREADS;
	m_threads10.y = NUM_THREADS;
	m_threads10.z = 10;
	m_threads5.x = NUM_THREADS;
	m_threads5.y = NUM_THREADS;
	m_threads5.z = 5;
	m_threads2.x = NUM_THREADS;
	m_threads2.y = NUM_THREADS;
	m_threads2.z = 2;
	m_threads1.x = NUM_THREADS;
	m_threads1.y = NUM_THREADS;
	m_threads1.z = 1;
	m_grid.x = static_cast<int>(fmax(ceil(dimX / static_cast<double>(NUM_THREADS)), 1.0));
	m_grid.y = static_cast<int>(fmax(ceil(dimY / static_cast<double>(NUM_THREADS)), 1.0));
	m_frame1.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_frame2.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_blurredFrame1.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_blurredFrame2.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_imageDeltaArray.init({5, 2, dimY, dimX});
	m_offsetArray12.init({2, 5, dimY, dimX});
	m_offsetArray21.init({2, dimY, dimX});
	m_blurredOffsetArray12.init({2, dimY, dimX});
	m_blurredOffsetArray21.init({2, dimY, dimX});
	m_statusArray.init({dimY, dimX});
	m_summedUpDeltaArray.init({5, dimY, dimX});
	m_normalizedDeltaArray.init({5, dimY, dimX});
	m_lowestLayerArray.init({dimY, dimX});
	m_warpedFrame12.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_warpedFrame21.init({1, dimY, dimX}, 0, static_cast<size_t>(1.5 * static_cast<double>(dimY * dimX)));
	m_outputFrame.init({1, dimY, dimX}, 0, static_cast<size_t>(3.0 * dimY * dimX * dDimScalar));
	m_hitCount12.init({1, dimY, dimX});
	m_hitCount21.init({1, dimY, dimX});
	m_ones.init({1, dimY, dimX}, 1);
}

/*
* Updates the frame1 array
*
* @param pInBuffer: Pointer to the input frame
*/
void OpticalFlowCalcSDR::updateFrame1(const unsigned char* pInBuffer) {
	m_frame1.fillData(pInBuffer);
	m_bBisNewest = false;
}

/*
* Updates the frame2 array
*
* @param pInBuffer: Pointer to the input frame
*/
void OpticalFlowCalcSDR::updateFrame2(const unsigned char* pInBuffer) {
	m_frame2.fillData(pInBuffer);
	m_bBisNewest = true;
}

/*
* Copies the frame in the correct format to the output buffer
*
* @param pInBuffer: Pointer to the input frame
* @param pOutBuffer: Pointer to the output frame
*/
void OpticalFlowCalcSDR::copyFrame(const unsigned char* pInBuffer, unsigned char* pOutBuffer) {
	// Set the array entries to the provided value
	m_frame1.fillData(pInBuffer);

	// Convert the NV12 frame to P010
	convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}

	// Download the output frame
	m_outputFrame.download(pOutBuffer);
}

/*
* Blurs a frame
*
* @param kernelSize: Size of the kernel to use for the blur
* @param directOutput: Whether to output the blurred frame directly
*/
void OpticalFlowCalcSDR::blurFrameArray(const int kernelSize, const bool directOutput) {
	if (!m_bBisNewest) {
		// Launch kernel
		blurFrameKernel << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_blurredFrame1.arrayPtrGPU, kernelSize, m_iDimY, m_iDimX);

		// Convert the NV12 frame to P010 if we are doing direct output
		if (directOutput) {
			convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_blurredFrame1.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		}
	} else {
		// Launch kernel
		blurFrameKernel << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_blurredFrame2.arrayPtrGPU, kernelSize, m_iDimY, m_iDimX);

		// Convert the NV12 frame to P010 if we are doing direct output
		if (directOutput) {
			convertNV12toP010KernelSDR << <m_grid, m_threads2 >> > (m_blurredFrame2.arrayPtrGPU, m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		}
	}

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}

/*
* Calculates the optical flow between frame1 and frame2
*
* @param iNumIterations: Number of iterations to calculate the optical flow
* @param iNumSteps: Number of steps executed to find the ideal offset (limits the maximum offset)
*/
void OpticalFlowCalcSDR::calculateOpticalFlow(unsigned int iNumIterations, unsigned int iNumSteps) {
	// Reset variables
	const int directionIdxOffset = m_iNumLayers * m_iLowDimY * m_iLowDimX; // Offset to index the Y-Offset-Layer
	const int channelIdxOffset = m_iDimY * m_iDimX; // Offset to index the color channel of the current thread
	unsigned int windowDimX = m_iLowDimX;
	unsigned int windowDimY = m_iLowDimY;
	if (iNumIterations == 0 || static_cast<float>(iNumIterations) > ceil(log2f(static_cast<float>(m_iLowDimX)))) {
		iNumIterations = static_cast<unsigned int>(ceil(log2f(static_cast<float>(m_iLowDimX))));
	}

	// Set the starting offset for the current window size
	setInitialOffset << <m_lowGrid, m_threads5 >> > (m_offsetArray12.arrayPtrGPU, m_iNumLayers, m_iLowDimY, m_iLowDimX);

	// We calculate the ideal offset array for each window size (entire frame, ..., individual pixels)
	for (unsigned int iter = 0; iter < iNumIterations; iter++) {
		// Each step we adjust the offset array to find the ideal offset
		for (unsigned int step = 0; step < iNumSteps; step++) {
			// Reset the summed up delta array
			m_summedUpDeltaArray.zero();

			// 1. Calculate the image deltas with the current offset array
			if (m_bBisNewest) {
				calcImageDeltaSDR << <m_lowGrid, m_threads10 >> > (m_blurredFrame1.arrayPtrGPU, m_blurredFrame2.arrayPtrGPU,
															   m_imageDeltaArray.arrayPtrGPU, m_offsetArray12.arrayPtrGPU,
															   m_iNumLayers, m_iLowDimY, m_iLowDimX, m_iDimY, m_iDimX,
															   m_dResolutionScalar, directionIdxOffset, channelIdxOffset);
			} else {
				calcImageDeltaSDR << <m_lowGrid, m_threads10 >> > (m_blurredFrame2.arrayPtrGPU, m_blurredFrame1.arrayPtrGPU,
															   m_imageDeltaArray.arrayPtrGPU, m_offsetArray12.arrayPtrGPU,
															   m_iNumLayers, m_iLowDimY, m_iLowDimX, m_iDimY, m_iDimX,
															   m_dResolutionScalar, directionIdxOffset, channelIdxOffset);
			}

			// 2. Sum up the deltas of each window
			calcDeltaSumsSDR << <m_lowGrid, m_threads10 >> > (m_imageDeltaArray.arrayPtrGPU, m_summedUpDeltaArray.arrayPtrGPU,
														  windowDimY, windowDimX, m_iNumLayers, m_iLowDimY, m_iLowDimX);

			// 3. Normalize the summed up delta array and find the best layer
			normalizeDeltaSums << <m_lowGrid, m_threads5 >> > (m_summedUpDeltaArray.arrayPtrGPU, m_lowestLayerArray.arrayPtrGPU,
															   m_offsetArray12.arrayPtrGPU, windowDimY, windowDimX,
															   m_iNumLayers, m_iLowDimY, m_iLowDimX);

			// 4. Adjust the offset array based on the comparison results
			adjustOffsetArray << <m_lowGrid, m_threads1 >> > (m_offsetArray12.arrayPtrGPU, m_lowestLayerArray.arrayPtrGPU,
															  m_statusArray.arrayPtrGPU, windowDimY, windowDimX, 
															  m_iNumLayers, m_iLowDimY, m_iLowDimX);
		}

		// 5. Adjust window size
		windowDimX = max(windowDimX / 2, 1);
		windowDimY = max(windowDimY / 2, 1);

		// Reset the status array
		m_statusArray.zero();
	}

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}

/*
* Warps the frames according to the calculated optical flow
*
* @param fScalar: The scalar to blend the frames with
* @param bOutput12: Whether to output the warped frame 12 or 21
*/
void OpticalFlowCalcSDR::warpFramesForOutput(float fScalar, const bool bOutput12) {
	// Calculate the blend scalar
	const float frameScalar12 = fScalar;
	const float frameScalar21 = static_cast<float>(1.0) - fScalar;

	// Calculate variables so the threds don't have to do it
	const int directionIdxOffset = m_iDimY * m_dResolutionDivider * m_iDimX * m_dResolutionDivider;
	const int scaledDimX = static_cast<unsigned int>(m_iDimX * m_dDimScalar);
	const int channelIdxOffset = static_cast<unsigned int>(m_iDimY * m_iDimX * m_dDimScalar);

	// Reset the hit count array
	if (bOutput12) {
		m_hitCount12.zero();
	} else {
		m_hitCount21.zero();
	}

	// Launch kernels
	if (m_bBisNewest) {
		// Frame 1 to Frame 2
		if (bOutput12) {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
														          m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_outputFrame.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		// Frame 2 to Frame 1
		} else {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU, 
																  m_outputFrame.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		}
	} else {
		// Frame 1 to Frame 2
		if (bOutput12) {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
																  m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_outputFrame.arrayPtrGPU, frameScalar12, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame2.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																		m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		// Frame 2 to Frame 1
		} else {
			warpFrameKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																  m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU,
																  m_outputFrame.arrayPtrGPU, frameScalar21, m_iLowDimY, m_iLowDimX,
																  m_iDimY, m_iDimX, m_dResolutionDivider, directionIdxOffset, scaledDimX, channelIdxOffset);
			artifactRemovalKernelForOutputSDR << <m_grid, m_threads2 >> > (m_frame1.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																	    m_outputFrame.arrayPtrGPU, m_iDimY, m_iDimX, m_dDimScalar);
		}
	}

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}

/*
* Warps the frames according to the calculated optical flow
*
* @param fScalar: The scalar to blend the frames with
*/
void OpticalFlowCalcSDR::warpFramesForBlending(float fScalar) {
	// Calculate the blend scalar
	const float frameScalar12 = fScalar;
	const float frameScalar21 = static_cast<float>(1.0) - fScalar;

	// Reset the hit count array
	m_hitCount12.zero();
	m_hitCount21.zero();

	// Create CUDA streams
	hipStream_t warpStream1, warpStream2;
	hipStreamCreate(&warpStream1);
	hipStreamCreate(&warpStream2);

	// Launch kernels
	if (m_bBisNewest) {
		// Frame 1 to Frame 2
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
																			    m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU, 
																			    m_warpedFrame12.arrayPtrGPU, frameScalar12, m_iDimY, 
																			    m_iDimX, m_dResolutionDivider);
		artifactRemovalKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame1.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																				      m_warpedFrame12.arrayPtrGPU, m_iDimY, m_iDimX);

		// Frame 2 to Frame 1
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																			    m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU, 
																				m_warpedFrame21.arrayPtrGPU, frameScalar21, m_iDimY, 
																				m_iDimX, m_dResolutionDivider);
		artifactRemovalKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame2.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																					  m_warpedFrame21.arrayPtrGPU, m_iDimY, m_iDimX);

	} else {
		// Frame 1 to Frame 2
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame2.arrayPtrGPU, m_blurredOffsetArray12.arrayPtrGPU,
																				m_hitCount12.arrayPtrGPU, m_ones.arrayPtrGPU,
																				m_warpedFrame12.arrayPtrGPU, frameScalar12, m_iDimY,
																				m_iDimX, m_dResolutionDivider);
		artifactRemovalKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream1 >> > (m_frame2.arrayPtrGPU, m_hitCount12.arrayPtrGPU,
																					  m_warpedFrame12.arrayPtrGPU, m_iDimY, m_iDimX);

		// Frame 2 to Frame 1
		warpFrameKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame1.arrayPtrGPU, m_blurredOffsetArray21.arrayPtrGPU,
																				m_hitCount21.arrayPtrGPU, m_ones.arrayPtrGPU,
																				m_warpedFrame21.arrayPtrGPU, frameScalar21, m_iDimY,
																				m_iDimX, m_dResolutionDivider);
		artifactRemovalKernelForBlendingSDR << <m_grid, m_threads2, 0, warpStream2 >> > (m_frame1.arrayPtrGPU, m_hitCount21.arrayPtrGPU,
																					  m_warpedFrame21.arrayPtrGPU, m_iDimY, m_iDimX);

	}

	// Synchronize streams to ensure completion
	hipStreamSynchronize(warpStream1);
	hipStreamSynchronize(warpStream2);

	// Clean up streams
	hipStreamDestroy(warpStream1);
	hipStreamDestroy(warpStream2);

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}

/*
* Blends warpedFrame1 to warpedFrame2
*
* @param dScalar: The scalar to blend the frames with
*/
void OpticalFlowCalcSDR::blendFrames(float fScalar) {
	// Calculate the blend scalar
	const float frame1Scalar = static_cast<float>(1.0) - fScalar;
	const float frame2Scalar = fScalar;

	// Blend the frames
	blendFrameKernelSDR << <m_grid, m_threads2 >> >(m_warpedFrame12.arrayPtrGPU, m_warpedFrame21.arrayPtrGPU,
												 m_outputFrame.arrayPtrGPU, frame1Scalar, frame2Scalar,
	                                             m_iDimY, m_iDimX, m_dDimScalar);

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}

/*
* Draws the flow as an RGB image
*
* @param saturation: The saturation of the flow image
* @param value: The value of the flow image
*/
void OpticalFlowCalcSDR::drawFlowAsHSV(const float saturation, const float value) const {
	if (m_bBisNewest) {
		convertFlowToHSVKernelSDR << <m_grid, m_threads2 >> > (m_blurredOffsetArray12.arrayPtrGPU, m_outputFrame.arrayPtrGPU,
														    m_frame2.arrayPtrGPU, m_iDimY, m_iDimX, 
															saturation, value, m_dResolutionDivider, m_dDimScalar);
	} else {
		convertFlowToHSVKernelSDR << <m_grid, m_threads2 >> > (m_blurredOffsetArray12.arrayPtrGPU, m_outputFrame.arrayPtrGPU,
														    m_frame1.arrayPtrGPU, m_iDimY, m_iDimX, 
															saturation, value, m_dResolutionDivider, m_dDimScalar);
	}

	// Check for CUDA errors
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(hipError_t));
		exit(-1);
	}
}